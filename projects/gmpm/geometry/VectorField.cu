#include "hip/hip_runtime.h"
#include "zensim/Logger.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/omp/execution/ExecutionPolicy.hpp"
#include "zensim/geometry/PoissonDisk.hpp"
#include "zensim/geometry/VdbLevelSet.h"
#include "zensim/geometry/VdbSampler.h"
#include "zensim/io/MeshIO.hpp"
#include "zensim/math/bit/Bits.h"
#include "zensim/types/Property.h"
#include <atomic>
#include <zeno/VDBGrid.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/StringObject.h>

#include "kernel/gradient_field.hpp"

namespace zeno {

struct ZSEvalGradientField : zeno::INode {
    using T = float;
    using dtiles_t = zs::TileVector<T,32>;
    using tiles_t = typename ZenoParticles::particles_t;
    using vec3 = zs::vec<T,3>;
    using mat3 = zs::vec<T,3,3>;

    virtual void apply() override {
        using namespace zs;
        auto field = get_input<ZenoParticles>("field");
        auto& verts = field->getParticles();

        auto attr = get_param<std::string>("tag");
        auto attrg = get_param<std::string>("gtag");
        if(!verts.hasProperty(attr)){
            fmt::print("the input field does not contain specified channel:{}\n",attr);
            throw std::runtime_error("the input field does not contain specified channel");
        }
        if(verts.getChannelSize(attr) != 1){
            fmt::print("only scaler field is currently supported\n");
            throw std::runtime_error("only scaler field is currently supported");
        }

        auto& eles = field->getQuadraturePoints();
        auto simplex_size = eles.getChannelSize("inds");
        if(simplex_size != 4 && simplex_size != 3)
            throw std::runtime_error("ZSEvalGradientField: invalid simplex size");

        static dtiles_t etemp(eles.get_allocator(),{{"g",3}},eles.size());
        static dtiles_t vtemp{verts.get_allocator(),{
            {"T",1},
        },verts.size()};

        etemp.resize(eles.size());
        vtemp.resize(verts.size());

        constexpr auto space = execspace_e::cuda;
        auto cudaPol = cuda_exec();

        eles.append_channels(cudaPol,{{attrg,3}});
        // copy the scaler field from verts to vtemp
        cudaPol(zs::range(verts.size()),
            [verts = proxy<space>({},verts),vtemp = proxy<space>({},vtemp),attr = zs::SmallString(attr),tag = zs::SmallString("T")]
                ZS_LAMBDA(int vi) mutable {
                    vtemp(tag,vi) = verts(attr,vi);
        });
        // compute_gradient(cudaPol,eles,verts,"x",vtemp,"T",etemp,"g",zs::wrapv(simplex_size));
        if(simplex_size == 4)
            compute_gradient<4>(cudaPol,eles,verts,"x",vtemp,"T",etemp,"g");
        if(simplex_size == 3)
            compute_gradient<3>(cudaPol,eles,verts,"x",vtemp,"T",etemp,"g");
        // copy the gradient field from etemp to eles
        cudaPol(zs::range(eles.size()),
            [eles = proxy<space>({},eles),etemp = proxy<space>({},etemp),gtag = zs::SmallString(attrg)]
                ZS_LAMBDA(int ei) mutable {
                    eles.tuple<3>(gtag,ei) = etemp.pack<3>("g",ei);
        });
        set_output("field",field);
    }
};

ZENDEFNODE(ZSEvalGradientField, {
                                    {"field"},
                                    {"field"},
                                    {
                                        {"string","tag","T"},{"string","gtag","gradT"}
                                    },
                                    {"ZSGeometry"}
});

struct ZSRetrieveVectorField : zeno::INode {
    using T = float;
    using dtiles_t = zs::TileVector<T,32>;
    using tiles_t = typename ZenoParticles::particles_t;
    using vec3 = zs::vec<T,3>;
    using mat3 = zs::vec<T,3,3>;
    virtual void apply() override {
        using namespace zs;
        auto field = get_input<ZenoParticles>("field");
        const auto& verts = field->getParticles();
        const auto& eles = field->getQuadraturePoints(); 

        auto type = get_param<std::string>("location");
        auto gtag = get_param<std::string>("gtag");
        auto xtag = get_param<std::string>("xtag");
        auto normalize = get_param<int>("normalize");
        auto scale = (T)get_param<float>("scale");

        if(type == "element" && !eles.hasProperty(gtag)){
            fmt::print("the volume does not contain element-wise gradient field : {}\n",gtag);
            throw std::runtime_error("the volume does not contain element-wise gradient field");
        }
        if(type == "vert" && !verts.hasProperty(gtag)){
            fmt::print("the volume does not contain nodal-wize gradient field : {}\n",gtag);
            throw std::runtime_error("the volume does not contain nodal-wize gradient field");
        }
        if(!verts.hasProperty(xtag)){
            fmt::print("the volume does not contain specified position channel : {}\n",xtag);
            throw std::runtime_error("the volume does not contain specified position channel");
        }

        std::vector<zs::PropertyTag> tags{{"x",3},{"vec",3}/*,{"clr",3}*/};
        bool on_elm = (type == "element");
        auto vec_buffer = typename ZenoParticles::particles_t(tags,on_elm ? eles.size() : verts.size(),zs::memsrc_e::device,0);
        // transfer the data from gpu to cpu
        constexpr auto cuda_space = execspace_e::cuda;
        auto cudaPol = cuda_exec();

        int elm_dim = eles.getChannelSize("inds");

        cudaPol(zs::range(vec_buffer.size()),
            [vec_buffer = proxy<cuda_space>({},vec_buffer),verts = proxy<cuda_space>({},verts),eles = proxy<cuda_space>({},eles),
                gtag = zs::SmallString(gtag),xtag = zs::SmallString(xtag),on_elm,scale,normalize,elm_dim] ZS_LAMBDA(int i) mutable {
                    if(on_elm){
                        auto bx = vec3::zeros();
                        if(elm_dim == 4){
                            auto inds = eles.pack<4>("inds",i).reinterpret_bits<int>();
                            for(int j = 0;j != 4;++j)
                                bx += verts.pack<3>(xtag,inds[j]) / 4;
                        }else if(elm_dim == 3){
                            auto inds = eles.pack<3>("inds",i).reinterpret_bits<int>();
                            for(int j= 0;j != 3;++j)
                                bx += verts.pack<3>("xtag",inds[j]) / 3;
                        }
                        vec_buffer.tuple<3>("x",i) = bx;
                        vec_buffer.tuple<3>("vec",i) = scale * eles.pack<3>(gtag,i) / eles.pack<3>(gtag,i).norm();
                    }else{
                        vec_buffer.tuple<3>("x",i) = verts.pack<3>(xtag,i);
                        vec_buffer.tuple<3>("vec",i) = scale * verts.pack<3>(gtag,i) / verts.pack<3>(gtag,i).norm();
                    }
        });

        vec_buffer = vec_buffer.clone({zs::memsrc_e::host});
        int vec_size = vec_buffer.size();
        constexpr auto omp_space = execspace_e::openmp;
        auto ompPol = omp_exec();

        auto vec_field = std::make_shared<zeno::PrimitiveObject>();
        vec_field->resize(vec_size * 2);
        auto& segs = vec_field->lines;
        segs.resize(vec_size);
        auto& sverts = vec_field->attr<zeno::vec3f>("pos");

        ompPol(zs::range(vec_buffer.size()),
            [vec_buffer = proxy<omp_space>({},vec_buffer),&segs,&sverts,vec_size] (int i) mutable {
                segs[i] = zeno::vec2i(i * 2 + 0,i * 2 + 1);
                auto start = vec_buffer.pack<3>("x",i);
                auto end = start + vec_buffer.pack<3>("vec",i);
                sverts[i*2 + 0] = zeno::vec3f{start[0],start[1],start[2]};
                sverts[i*2 + 1] = zeno::vec3f{end[0],end[1],end[2]};
        });

        set_output("vec_field",std::move(vec_field));
    }    
};

ZENDEFNODE(ZSRetrieveVectorField, {
    {"field"},
    {"vec_field"},
    {{"enum element vert","location","element"},{"string","gtag","vec_field"},{"string","xtag","xtag"},{"float","scale","1.0"},{"int","normalize","1"}},
    {"ZSGeometry"},
});


};