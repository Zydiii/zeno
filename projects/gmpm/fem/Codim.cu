#include "hip/hip_runtime.h"
#include "../Structures.hpp"
#include "../Utils.hpp"
// #include "Ccd.hpp"
#include "Ccds.hpp"
#include "zensim/Logger.hpp"
#include "zensim/cuda/execution/ExecutionPolicy.cuh"
#include "zensim/execution/ExecutionPolicy.hpp"
#include "zensim/geometry/Distance.hpp"
#include "zensim/geometry/PoissonDisk.hpp"
#include "zensim/geometry/SpatialQuery.hpp"
#include "zensim/geometry/VdbLevelSet.h"
#include "zensim/geometry/VdbSampler.h"
#include "zensim/io/MeshIO.hpp"
#include "zensim/math/bit/Bits.h"
#include "zensim/physics/ConstitutiveModel.hpp"
#include "zensim/types/Property.h"
#include <atomic>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <zeno/VDBGrid.h>
#include <zeno/types/ListObject.h>
#include <zeno/types/NumericObject.h>
#include <zeno/types/PrimitiveObject.h>
#include <zeno/types/StringObject.h>

namespace zeno {

template <typename TileVecT, int codim = 3>
zs::Vector<zs::AABBBox<3, typename TileVecT::value_type>>
retrieve_bounding_volumes(zs::CudaExecutionPolicy &pol, const TileVecT &vtemp,
                          const zs::SmallString &xTag,
                          const typename ZenoParticles::particles_t &eles,
                          zs::wrapv<codim>, int voffset) {
  using namespace zs;
  using T = typename TileVecT::value_type;
  using bv_t = AABBBox<3, T>;
  static_assert(codim >= 1 && codim <= 4, "invalid co-dimension!\n");
  constexpr auto space = execspace_e::cuda;
  zs::Vector<bv_t> ret{eles.get_allocator(), eles.size()};
  pol(range(eles.size()), [eles = proxy<space>({}, eles),
                           bvs = proxy<space>(ret),
                           vtemp = proxy<space>({}, vtemp),
                           codim_v = wrapv<codim>{}, xTag,
                           voffset] ZS_LAMBDA(int ei) mutable {
    constexpr int dim = RM_CVREF_T(codim_v)::value;
    auto inds =
        eles.template pack<dim>("inds", ei).template reinterpret_bits<int>() +
        voffset;
    auto x0 = vtemp.template pack<3>(xTag, inds[0]);
    bv_t bv{x0, x0};
    for (int d = 1; d != dim; ++d)
      merge(bv, vtemp.template pack<3>(xTag, inds[d]));
    bvs[ei] = bv;
  });
  return ret;
}
template <typename TileVecT0, typename TileVecT1, int codim = 3>
zs::Vector<zs::AABBBox<3, typename TileVecT0::value_type>>
retrieve_bounding_volumes(zs::CudaExecutionPolicy &pol, const TileVecT0 &verts,
                          const zs::SmallString &xTag,
                          const typename ZenoParticles::particles_t &eles,
                          zs::wrapv<codim>, const TileVecT1 &vtemp,
                          const zs::SmallString &dirTag, float stepSize,
                          int voffset) {
  using namespace zs;
  using T = typename TileVecT0::value_type;
  using bv_t = AABBBox<3, T>;
  static_assert(codim >= 1 && codim <= 4, "invalid co-dimension!\n");
  constexpr auto space = execspace_e::cuda;
  Vector<bv_t> ret{eles.get_allocator(), eles.size()};
  pol(zs::range(eles.size()), [eles = proxy<space>({}, eles),
                               bvs = proxy<space>(ret),
                               verts = proxy<space>({}, verts),
                               vtemp = proxy<space>({}, vtemp),
                               codim_v = wrapv<codim>{}, xTag, dirTag, stepSize,
                               voffset] ZS_LAMBDA(int ei) mutable {
    constexpr int dim = RM_CVREF_T(codim_v)::value;
    auto inds =
        eles.template pack<dim>("inds", ei).template reinterpret_bits<int>() +
        voffset;
    auto x0 = verts.template pack<3>(xTag, inds[0]);
    auto dir0 = vtemp.template pack<3>(dirTag, inds[0]);
    bv_t bv{get_bounding_box(x0, x0 + stepSize * dir0)};
    for (int d = 1; d != dim; ++d) {
      auto x = verts.template pack<3>(xTag, inds[d]);
      auto dir = vtemp.template pack<3>(dirTag, inds[d]);
      merge(bv, x);
      merge(bv, x + stepSize * dir);
    }
    bvs[ei] = bv;
  });
  return ret;
}

struct CodimStepping : INode {
  using T = double;
  using Ti = zs::conditional_t<zs::is_same_v<T, double>, zs::i64, zs::i32>;
  using dtiles_t = zs::TileVector<T, 32>;
  using tiles_t = typename ZenoParticles::particles_t;
  using vec3 = zs::vec<T, 3>;
  using ivec3 = zs::vec<int, 3>;
  using ivec2 = zs::vec<int, 2>;
  using mat2 = zs::vec<T, 2, 2>;
  using mat3 = zs::vec<T, 3, 3>;
  using pair_t = zs::vec<int, 2>;
  using pair3_t = zs::vec<int, 3>;
  using bvh_t = zs::LBvh<3, 32, int, T>;
  using bv_t = zs::AABBBox<3, T>;

  static constexpr vec3 s_groundNormal{0, 1, 0};
  inline static const char s_meanMassTag[] = "MeanMass";
  inline static const char s_meanSurfEdgeLengthTag[] = "MeanSurfEdgeLength";
  inline static int refStepsizeCoeff = 1;
  inline static int numContinuousCap = 0;
  inline static bool projectDBC = true;
  inline static bool BCsatisfied = false;
  inline static int PNCap = 1000;
  inline static int CGCap = 500;
  inline static int CCDCap = 20000;
  inline static T updateZoneTol = 1e-1;
  inline static T consTol = 1e-2;
  inline static T armijoParam = 1e-4;
  inline static bool useGD = false;
  inline static T boxDiagSize2 = 0;
  inline static T avgNodeMass = 0;
  inline static T targetGRes = 1e-2;
#define s_enableAdaptiveSetting 1
// static constexpr bool s_enableAdaptiveSetting = false;
#define s_enableContact 1
// static constexpr bool s_enableContact = true;
#define s_enableDCDCheck 0
  // static constexpr bool s_enableDCDCheck = false;

  inline static std::size_t estNumCps = 1000000;
  inline static T augLagCoeff = 1e4;
  inline static T cgRel = 1e-2;
  inline static T pnRel = 1e-2;
  inline static T kappaMax = 1e8;
  inline static T kappaMin = 1e4;
  inline static T kappa0 = 1e4;
  inline static T kappa = kappa0;
  inline static T xi = 0; // 1e-2; // 2e-3;
  inline static T dHat = 0.0025;
  inline static vec3 extForce;

  template <typename T> static inline T computeHb(const T d2, const T dHat2) {
#if 0
    T hess = 0;
    if (d2 < dHat2) {
      T t2 = d2 - dHat2;
      hess = (std::log(d2 / dHat2) * (T)-2.0 - t2 * (T)4.0 / d2) / (dHat2 * dHat2)
                + 1.0 / (d2 * d2) * (t2 / dHat2) * (t2 / dHat2);
    }
    return hess;
#else
    if (d2 >= dHat2)
      return 0;
    T t2 = d2 - dHat2;
    return ((std::log(d2 / dHat2) * -2 - t2 * 4 / d2) + (t2 / d2) * (t2 / d2));
#endif
  }

  template <
      typename VecT, int N = VecT::template range_t<0>::value,
      zs::enable_if_all<N % 3 == 0, N == VecT::template range_t<1>::value> = 0>
  static constexpr void rotate_hessian(zs::VecInterface<VecT> &H,
                                       const mat3 BCbasis[N / 3],
                                       const int BCorder[N / 3],
                                       const int BCfixed[], bool projectDBC) {
    // hessian rotation: trans^T hess * trans
    // left trans^T: multiplied on rows
    // right trans: multiplied on cols
    constexpr int NV = N / 3;
    // rotate and project
    for (int vi = 0; vi != NV; ++vi) {
      int offsetI = vi * 3;
      for (int vj = 0; vj != NV; ++vj) {
        int offsetJ = vj * 3;
        mat3 tmp{};
        for (int i = 0; i != 3; ++i)
          for (int j = 0; j != 3; ++j)
            tmp(i, j) = H(offsetI + i, offsetJ + j);
        // rotate
        tmp = BCbasis[vi].transpose() * tmp * BCbasis[vj];
        // project
        if (projectDBC) {
          if (BCorder[vi] > 0 || BCorder[vj] > 0) {
            if (vi == vj) {
              for (int i = 0; i != BCorder[vi]; ++i)
                for (int j = 0; j != BCorder[vj]; ++j)
                  tmp(i, j) = (i == j ? 1 : 0);
            } else {
              for (int i = 0; i != BCorder[vi]; ++i)
                for (int j = 0; j != BCorder[vj]; ++j)
                  tmp(i, j) = 0;
            }
          }
        } else {
          if (BCorder[vi] > 0 && BCfixed[vi] ||
              BCorder[vj] > 0 && BCfixed[vj]) {
            if (vi == vj) {
              for (int i = 0; i != BCorder[vi]; ++i)
                for (int j = 0; j != BCorder[vj]; ++j)
                  tmp(i, j) = (i == j ? 1 : 0);
            } else {
              for (int i = 0; i != BCorder[vi]; ++i)
                for (int j = 0; j != BCorder[vj]; ++j)
                  tmp(i, j) = 0;
            }
          }
        }
        for (int i = 0; i != 3; ++i)
          for (int j = 0; j != 3; ++j)
            H(offsetI + i, offsetJ + j) = tmp(i, j);
      }
    }
    return;
  }

  /// ref: codim-ipc
  struct IPCSystem {

    /// utilities
    static constexpr std::size_t count_warps(std::size_t n) noexcept {
      return (n + 31) / 32;
    }
    static constexpr int warp_index(int n) noexcept { return n / 32; }
    static constexpr auto warp_mask(int i, int n) noexcept {
      int k = n % 32;
      const int tail = n - k;
      if (i < tail)
        return zs::make_tuple(0xFFFFFFFFu, 32);
      return zs::make_tuple(((unsigned)(1ull << k) - 1), k);
    }

    template <typename T>
    static __forceinline__ __device__ void reduce_to(int i, int n, T val,
                                                     T &dst) {
      auto [mask, numValid] = warp_mask(i, n);
      __syncwarp(mask);
      auto locid = threadIdx.x & 31;
      for (int stride = 1; stride < 32; stride <<= 1) {
        auto tmp = __shfl_down_sync(mask, val, stride);
        if (locid + stride < numValid)
          val += tmp;
      }
      if (locid == 0)
        zs::atomic_add(zs::exec_cuda, &dst, val);
    }

    void clearTemp(std::size_t size) {
      zs::memset(zs::mem_device, temp.data(), 0, sizeof(T) * size);
    }
    template <typename Op = std::plus<T>>
    T reduce(zs::CudaExecutionPolicy &cudaPol, const zs::Vector<T> &res,
             Op op = {}) {
      using namespace zs;
      Vector<T> ret{res.get_allocator(), 1};
      zs::reduce(cudaPol, std::begin(res), std::end(res), std::begin(ret), (T)0,
                 op);
      return ret.getVal();
    }
    T dot(zs::CudaExecutionPolicy &cudaPol, dtiles_t &vertData,
          const zs::SmallString tag0, const zs::SmallString tag1) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      // Vector<double> res{vertData.get_allocator(), vertData.size()};
      Vector<double> res{vertData.get_allocator(),
                         count_warps(vertData.size())};
      zs::memset(zs::mem_device, res.data(), 0,
                 sizeof(double) * count_warps(vertData.size()));
      cudaPol(range(vertData.size()),
              [data = proxy<space>({}, vertData), res = proxy<space>(res), tag0,
               tag1, n = vertData.size()] __device__(int pi) mutable {
                auto v0 = data.pack<3>(tag0, pi);
                auto v1 = data.pack<3>(tag1, pi);
                auto v = v0.dot(v1);
                // res[pi] = v;
                reduce_to(pi, n, v, res[pi / 32]);
              });
      return reduce(cudaPol, res, std::plus<double>{});
    }
    T infNorm(zs::CudaExecutionPolicy &cudaPol, dtiles_t &vertData,
              const zs::SmallString tag = "dir") {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      Vector<T> res{vertData.get_allocator(), count_warps(vertData.size())};
      zs::memset(zs::mem_device, res.data(), 0,
                 sizeof(T) * count_warps(vertData.size()));
      cudaPol(range(vertData.size()),
              [data = proxy<space>({}, vertData), res = proxy<space>(res), tag,
               n = vertData.size()] __device__(int pi) mutable {
                auto v = data.pack<3>(tag, pi);
                auto val = v.abs().max();

                auto [mask, numValid] = warp_mask(pi, n);
                auto locid = threadIdx.x & 31;
                for (int stride = 1; stride < 32; stride <<= 1) {
                  auto tmp = __shfl_down_sync(mask, val, stride);
                  if (locid + stride < numValid)
                    val = zs::max(val, tmp);
                }
                if (locid == 0)
                  res[pi / 32] = val;
              });
      return reduce(cudaPol, res, getmax<T>{});
    }

    struct PrimitiveHandle {
      PrimitiveHandle(ZenoParticles &zsprim, std::size_t &vOffset,
                      std::size_t &sfOffset, std::size_t &seOffset,
                      std::size_t &svOffset, zs::wrapv<3>)
          : zsprim{zsprim}, verts{zsprim.getParticles<true>()},
            eles{zsprim.getQuadraturePoints()},
            etemp{zsprim.getQuadraturePoints().get_allocator(),
                  {{"He", 9 * 9}},
                  zsprim.numElements()},
            surfTris{zsprim.getQuadraturePoints()},
            surfEdges{zsprim[ZenoParticles::s_surfEdgeTag]},
            surfVerts{zsprim[ZenoParticles::s_surfVertTag]}, vOffset{vOffset},
            sfOffset{sfOffset}, seOffset{seOffset}, svOffset{svOffset},
            category{zsprim.category} {
        if (category != ZenoParticles::surface)
          throw std::runtime_error("dimension of 3 but is not surface");
        vOffset += verts.size();
        sfOffset += surfTris.size();
        seOffset += surfEdges.size();
        svOffset += surfVerts.size();
      }
      PrimitiveHandle(ZenoParticles &zsprim, std::size_t &vOffset,
                      std::size_t &sfOffset, std::size_t &seOffset,
                      std::size_t &svOffset, zs::wrapv<4>)
          : zsprim{zsprim}, verts{zsprim.getParticles<true>()},
            eles{zsprim.getQuadraturePoints()},
            etemp{zsprim.getQuadraturePoints().get_allocator(),
                  {{"He", 12 * 12}},
                  zsprim.numElements()},
            surfTris{zsprim[ZenoParticles::s_surfTriTag]},
            surfEdges{zsprim[ZenoParticles::s_surfEdgeTag]},
            surfVerts{zsprim[ZenoParticles::s_surfVertTag]}, vOffset{vOffset},
            sfOffset{sfOffset}, seOffset{seOffset}, svOffset{svOffset},
            category{zsprim.category} {
        if (category != ZenoParticles::tet)
          throw std::runtime_error("dimension of 4 but is not tetrahedra");
        vOffset += verts.size();
        sfOffset += surfTris.size();
        seOffset += surfEdges.size();
        svOffset += surfVerts.size();
      }

      T averageSurfEdgeLength(zs::CudaExecutionPolicy &pol,
                              IPCSystem &system) const {
        using namespace zs;
        constexpr auto space = execspace_e::cuda;
        if (zsprim.hasMeta(s_meanSurfEdgeLengthTag))
          return zsprim.readMeta(s_meanSurfEdgeLengthTag, zs::wrapt<T>{});
        auto &edges = surfEdges;
        Vector<T> edgeLengths{edges.get_allocator(), edges.size()};
        pol(Collapse{edges.size()},
            [edges = proxy<space>({}, edges), verts = proxy<space>({}, verts),
             edgeLengths =
                 proxy<space>(edgeLengths)] __device__(int ei) mutable {
              auto inds = edges.template pack<2>("inds", ei)
                              .template reinterpret_bits<int>();
              edgeLengths[ei] =
                  (verts.pack<3>("x0", inds[0]) - verts.pack<3>("x0", inds[1]))
                      .norm();
            });
        auto tmp = system.reduce(pol, edgeLengths) / edges.size();
        zsprim.setMeta(s_meanSurfEdgeLengthTag, tmp);
        return tmp;
      }

      decltype(auto) getVerts() const { return verts; }
      decltype(auto) getEles() const { return eles; }
      decltype(auto) getSurfTris() const { return surfTris; }
      decltype(auto) getSurfEdges() const { return surfEdges; }
      decltype(auto) getSurfVerts() const { return surfVerts; }

      ZenoParticles &zsprim;
      typename ZenoParticles::dtiles_t &verts;
      typename ZenoParticles::particles_t &eles;
      typename ZenoParticles::dtiles_t etemp;
      typename ZenoParticles::particles_t &surfTris;
      typename ZenoParticles::particles_t &surfEdges;
      // not required for codim obj
      typename ZenoParticles::particles_t &surfVerts;
      const std::size_t vOffset, sfOffset, seOffset, svOffset;
      ZenoParticles::category_e category;
    };

    T averageSurfEdgeLength(zs::CudaExecutionPolicy &pol) {
      T sumSurfEdgeLengths = 0;
      std::size_t sumSE = 0;
      for (auto &&primHandle : prims) {
        auto numSE = primHandle.getSurfEdges().size();
        sumSE += numSE;
        sumSurfEdgeLengths +=
            primHandle.averageSurfEdgeLength(pol, *this) * numSE;
      }
      return sumSurfEdgeLengths / sumSE;
    }

    ///
    auto getCnts() const {
      return zs::make_tuple(nPP.getVal(), nPE.getVal(), nPT.getVal(),
                            nEE.getVal(), ncsPT.getVal(), ncsEE.getVal());
    }
    void computeConstraints(zs::CudaExecutionPolicy &pol,
                            const zs::SmallString &tag) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      pol(Collapse{numDofs},
          [vtemp = proxy<space>({}, vtemp), tag] __device__(int vi) mutable {
            auto BCbasis = vtemp.pack<3, 3>("BCbasis", vi);
            auto BCtarget = vtemp.pack<3>("BCtarget", vi);
            int BCorder = vtemp("BCorder", vi);
            auto x = BCbasis.transpose() * vtemp.pack<3>(tag, vi);
            int d = 0;
            for (; d != BCorder; ++d)
              vtemp("cons", d, vi) = x[d] - BCtarget[d];
            for (; d != 3; ++d)
              vtemp("cons", d, vi) = 0;
          });
    }
    bool areConstraintsSatisfied(zs::CudaExecutionPolicy &pol) {
      using namespace zs;
      computeConstraints(pol, "xn");
      // auto res = infNorm(pol, vtemp, "cons");
      auto res = constraintResidual(pol);
      return res < 1e-2;
    }
    T checkDBCStatus(zs::CudaExecutionPolicy &pol) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      pol(Collapse{numDofs},
          [vtemp = proxy<space>({}, vtemp)] __device__(int vi) mutable {
            int BCorder = vtemp("BCorder", vi);
            if (BCorder > 0) {
              auto BCbasis = vtemp.pack<3, 3>("BCbasis", vi);
              auto BCtarget = vtemp.pack<3>("BCtarget", vi);
              auto cons = vtemp.pack<3>("cons", vi);
              auto xt = vtemp.pack<3>("xt", vi);
              auto x = vtemp.pack<3>("xn", vi);
              printf("%d-th vert (order [%d]): cur (%f, %f, %f) xt (%f, %f, %f)"
                     "\n\ttar(%f, %f, %f) cons (%f, %f, %f)\n",
                     vi, BCorder, (float)x[0], (float)x[1], (float)x[2],
                     (float)xt[0], (float)xt[1], (float)xt[2],
                     (float)BCtarget[0], (float)BCtarget[1], (float)BCtarget[2],
                     (float)cons[0], (float)cons[1], (float)cons[2]);
            }
          });
    }
    T constraintResidual(zs::CudaExecutionPolicy &pol,
                         bool maintainFixed = false) {
      using namespace zs;
      if (projectDBC)
        return 0;
      Vector<T> num{vtemp.get_allocator(), numDofs},
          den{vtemp.get_allocator(), numDofs};
      constexpr auto space = execspace_e::cuda;
      pol(Collapse{numDofs},
          [vtemp = proxy<space>({}, vtemp), den = proxy<space>(den),
           num = proxy<space>(num), maintainFixed] __device__(int vi) mutable {
            auto BCbasis = vtemp.pack<3, 3>("BCbasis", vi);
            auto BCtarget = vtemp.pack<3>("BCtarget", vi);
            int BCorder = vtemp("BCorder", vi);
            auto cons = vtemp.pack<3>("cons", vi);
            auto xt = vtemp.pack<3>("xt", vi);
            T n = 0, d_ = 0;
            // https://ipc-sim.github.io/file/IPC-supplement-A-technical.pdf Eq5
            for (int d = 0; d != BCorder; ++d) {
              n += zs::sqr(cons[d]);
              d_ += zs::sqr(col(BCbasis, d).dot(xt) - BCtarget[d]);
            }
            num[vi] = n;
            den[vi] = d_;
            if (maintainFixed && BCorder > 0) {
              if (d_ != 0) {
                if (zs::sqrt(n / d_) < 1e-6)
                  vtemp("BCfixed", vi) = 1;
              } else {
                if (zs::sqrt(n) < 1e-6)
                  vtemp("BCfixed", vi) = 1;
              }
            }
          });
      auto nsqr = reduce(pol, num);
      auto dsqr = reduce(pol, den);
      T ret = 0;
      if (dsqr == 0)
        ret = std::sqrt(nsqr);
      else
        ret = std::sqrt(nsqr / dsqr);
      return ret < 1e-6 ? 0 : ret;
    }
    void updateWholeBoundingBoxSize(zs::CudaExecutionPolicy &pol) const {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      Vector<bv_t> box{vtemp.get_allocator(), 1};
      pol(Collapse{1},
          [bvh = proxy<space>(stBvh), box = proxy<space>(box)] __device__(
              int vi) mutable { box[0] = bvh.getNodeBV(0); });
      bv_t bv = box.getVal();
      pol(Collapse{1},
          [bvh = proxy<space>(bouStBvh), box = proxy<space>(box)] __device__(
              int vi) mutable { box[0] = bvh.getNodeBV(0); });
      bv_t boubv = box.getVal();
      merge(bv, boubv._min);
      merge(bv, boubv._max);
      boxDiagSize2 = (bv._max - bv._min).l2NormSqr();
    }

    void computeInertialGradient(zs::CudaExecutionPolicy &cudaPol,
                                 const zs::SmallString &gTag) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      for (auto &primHandle : prims) {
        auto &verts = primHandle.getVerts();
        cudaPol(range(verts.size()),
                [vtemp = proxy<space>({}, vtemp),
                 verts = proxy<space>({}, verts), gTag, extForce = extForce,
                 dt = dt,
                 vOffset = primHandle.vOffset] __device__(int i) mutable {
                  auto m = verts("m", i);
                  int BCorder = vtemp("BCorder", vOffset + i);
                  if (BCorder != 3) {
                    // no need to neg
                    vtemp.tuple<3>(gTag, vOffset + i) =
                        -m * (vtemp.pack<3>("xn", vOffset + i) -
                              vtemp.pack<3>("xtilde", vOffset + i));
                  }
                });
      }
    }
    void initKappa(zs::CudaExecutionPolicy &pol) {
      // should be called after dHat set
      if (!s_enableContact)
        return;
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      pol(zs::range(numDofs),
          [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
            vtemp.tuple<3>("p", i) = vec3::zeros();
            vtemp.tuple<3>("q", i) = vec3::zeros();
          });
      // inertial + elasticity
      computeInertialGradient(pol, "p");
      match([&](auto &elasticModel) {
        computeElasticGradientAndHessian(pol, elasticModel, "p", false);
      })(models.getElasticModel());
      // contacts
      findCollisionConstraints(pol, dHat, xi, false);
      computeBarrierGradientAndHessian(pol, "q", false);
      computeBoundaryBarrierGradientAndHessian(pol, "q", false);

      auto gsum = dot(pol, vtemp, "p", "q");
      auto gsnorm = dot(pol, vtemp, "q", "q");
      if (gsnorm < limits<T>::min())
        kappaMin = 0;
      else
        kappaMin = -gsum / gsnorm;
      fmt::print("kappaMin: {}, gsum: {}, gsnorm: {}\n", kappaMin, gsum,
                 gsnorm);
    }
    bool updateKappaRequired(zs::CudaExecutionPolicy &pol) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      Vector<int> requireUpdate{vtemp.get_allocator(), 1};
      requireUpdate.setVal(0);
      // contacts
      {
        auto activeGap2 = dHat * dHat + 2 * xi * dHat;
        pol(range(prevNumPP),
            [vtemp = proxy<space>({}, vtemp), tempPP = proxy<space>({}, tempPP),
             requireUpdate = proxy<space>(requireUpdate),
             xi2 = xi * xi] __device__(int ppi) mutable {
              auto pp = tempPP.template pack<2>("inds_pre", ppi)
                            .template reinterpret_bits<i64>();
              auto x0 = vtemp.pack<3>("xn", pp[0]);
              auto x1 = vtemp.pack<3>("xn", pp[1]);
              auto dist2 = dist2_pp(x0, x1);
              if (dist2 - xi2 < tempPP("dist2_pre", ppi))
                requireUpdate[0] = 1;
            });
        pol(range(prevNumPE),
            [vtemp = proxy<space>({}, vtemp), tempPE = proxy<space>({}, tempPE),
             requireUpdate = proxy<space>(requireUpdate),
             xi2 = xi * xi] __device__(int pei) mutable {
              auto pe = tempPE.template pack<3>("inds_pre", pei)
                            .template reinterpret_bits<Ti>();
              auto p = vtemp.pack<3>("xn", pe[0]);
              auto e0 = vtemp.pack<3>("xn", pe[1]);
              auto e1 = vtemp.pack<3>("xn", pe[2]);
              auto dist2 = dist2_pe(p, e0, e1);
              if (dist2 - xi2 < tempPE("dist2_pre", pei))
                requireUpdate[0] = 1;
            });
        pol(range(prevNumPT),
            [vtemp = proxy<space>({}, vtemp), tempPT = proxy<space>({}, tempPT),
             requireUpdate = proxy<space>(requireUpdate),
             xi2 = xi * xi] __device__(int pti) mutable {
              auto pt = tempPT.template pack<4>("inds_pre", pti)
                            .template reinterpret_bits<Ti>();
              auto p = vtemp.pack<3>("xn", pt[0]);
              auto t0 = vtemp.pack<3>("xn", pt[1]);
              auto t1 = vtemp.pack<3>("xn", pt[2]);
              auto t2 = vtemp.pack<3>("xn", pt[3]);

              auto dist2 = dist2_pt(p, t0, t1, t2);
              if (dist2 - xi2 < tempPT("dist2_pre", pti))
                requireUpdate[0] = 1;
            });
        pol(range(prevNumEE),
            [vtemp = proxy<space>({}, vtemp), tempEE = proxy<space>({}, tempEE),
             requireUpdate = proxy<space>(requireUpdate),
             xi2 = xi * xi] __device__(int eei) mutable {
              auto ee = tempEE.template pack<4>("inds_pre", eei)
                            .template reinterpret_bits<Ti>();
              auto ea0 = vtemp.pack<3>("xn", ee[0]);
              auto ea1 = vtemp.pack<3>("xn", ee[1]);
              auto eb0 = vtemp.pack<3>("xn", ee[2]);
              auto eb1 = vtemp.pack<3>("xn", ee[3]);

              auto dist2 = dist2_ee(ea0, ea1, eb0, eb1);
              if (dist2 - xi2 < tempEE("dist2_pre", eei))
                requireUpdate[0] = 1;
            });
      }
      return requireUpdate.getVal();
    }

    void findCollisionConstraints(zs::CudaExecutionPolicy &pol, T dHat,
                                  T xi = 0, bool record = false) {
      nPP.setVal(0);
      nPE.setVal(0);
      nPT.setVal(0);
      nEE.setVal(0);

      ncsPT.setVal(0);
      ncsEE.setVal(0);
      {
        auto triBvs = retrieve_bounding_volumes(pol, vtemp, "xn", stInds,
                                                zs::wrapv<3>{}, 0);
        stBvh.refit(pol, triBvs);
        auto edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", seInds,
                                                 zs::wrapv<2>{}, 0);
        seBvh.refit(pol, edgeBvs);
      }
      findCollisionConstraintsImpl(pol, dHat, xi, false, record);

      {
        auto triBvs = retrieve_bounding_volumes(pol, vtemp, "xn", coEles,
                                                zs::wrapv<3>{}, coOffset);
        bouStBvh.refit(pol, triBvs);
        auto edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", coEdges,
                                                 zs::wrapv<2>{}, coOffset);
        bouSeBvh.refit(pol, edgeBvs);
      }
      findCollisionConstraintsImpl(pol, dHat, xi, true, record);

      if (record) {
        prevNumPP = nPP.getVal();
        prevNumPE = nPE.getVal();
        prevNumPT = nPT.getVal();
        prevNumEE = nEE.getVal();
      }
    }
    void findCollisionConstraintsImpl(zs::CudaExecutionPolicy &pol, T dHat,
                                      T xi, bool withBoundary = false,
                                      bool record = false) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      const auto dHat2 = dHat * dHat;

      /// pt
      pol(Collapse{svInds.size()},
          [svInds = proxy<space>({}, svInds),
           eles = proxy<space>({}, withBoundary ? coEles : stInds),
           vtemp = proxy<space>({}, vtemp),
           bvh = proxy<space>(withBoundary ? bouStBvh : stBvh),
           PP = proxy<space>(PP), nPP = proxy<space>(nPP),
           PE = proxy<space>(PE), nPE = proxy<space>(nPE),
           PT = proxy<space>(PT), nPT = proxy<space>(nPT),
           csPT = proxy<space>(csPT), ncsPT = proxy<space>(ncsPT), dHat, xi,
           thickness = xi + dHat, voffset = withBoundary ? coOffset : 0,
           record] __device__(int vi) mutable {
            vi = reinterpret_bits<int>(svInds("inds", vi));
            const auto dHat2 = zs::sqr(dHat + xi);
            int BCorder0 = vtemp("BCorder", vi);
            auto p = vtemp.template pack<3>("xn", vi);
            auto bv = bv_t{get_bounding_box(p - thickness, p + thickness)};
            bvh.iter_neighbors(bv, [&](int stI) {
              auto tri = eles.template pack<3>("inds", stI)
                             .template reinterpret_bits<int>() +
                         voffset;
              if (vi == tri[0] || vi == tri[1] || vi == tri[2])
                return;
              // all affected by sticky boundary conditions
              if (BCorder0 == 3 && vtemp("BCorder", tri[0]) == 3 &&
                  vtemp("BCorder", tri[1]) == 3 &&
                  vtemp("BCorder", tri[2]) == 3)
                return;
              // ccd
              auto t0 = vtemp.template pack<3>("xn", tri[0]);
              auto t1 = vtemp.template pack<3>("xn", tri[1]);
              auto t2 = vtemp.template pack<3>("xn", tri[2]);

              switch (pt_distance_type(p, t0, t1, t2)) {
              case 0: {
                if (auto d2 = dist2_pp(p, t0); d2 < dHat2) {
                  auto no = atomic_add(exec_cuda, &nPP[0], 1);
                  PP[no] = pair_t{vi, tri[0]};
                  csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] =
                      pair4_t{vi, tri[0], tri[1], tri[2]};

                  if (record) {
                    vtemp.template pack<2>("inds_pre", no) =
                        dpair_t{vi, tri[0]}.reinterpret_bits<T>();
                    vtemp("dist2_pre", no) = d2;
                  }
                }
                break;
              }
              case 1: {
                if (auto d2 = dist2_pp(p, t1); d2 < dHat2) {
                  auto no = atomic_add(exec_cuda, &nPP[0], 1);
                  PP[no] = pair_t{vi, tri[1]};
                  csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] =
                      pair4_t{vi, tri[0], tri[1], tri[2]};

                  if (record) {
                    vtemp.template pack<2>("inds_pre", no) =
                        dpair_t{vi, tri[1]}.reinterpret_bits<T>();
                    vtemp("dist2_pre", no) = d2;
                  }
                }
                break;
              }
              case 2: {
                if (auto d2 = dist2_pp(p, t2); d2 < dHat2) {
                  auto no = atomic_add(exec_cuda, &nPP[0], 1);
                  PP[no] = pair_t{vi, tri[2]};
                  csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] =
                      pair4_t{vi, tri[0], tri[1], tri[2]};

                  if (record) {
                    vtemp.template pack<2>("inds_pre", no) =
                        dpair_t{vi, tri[2]}.reinterpret_bits<T>();
                    vtemp("dist2_pre", no) = d2;
                  }
                }
                break;
              }
              case 3: {
                if (auto d2 = dist2_pe(p, t0, t1); d2 < dHat2) {
                  auto no = atomic_add(exec_cuda, &nPE[0], 1);
                  PE[no] = pair3_t{vi, tri[0], tri[1]};
                  csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] =
                      pair4_t{vi, tri[0], tri[1], tri[2]};
                  if (record) {
                    vtemp.template pack<3>("inds_pre", no) =
                        dpair3_t{vi, tri[0], tri[1]}.reinterpret_bits<T>();
                    vtemp("dist2_pre", no) = d2;
                  }
                }
                break;
              }
              case 4: {
                if (auto d2 = dist2_pe(p, t1, t2); d2 < dHat2) {
                  auto no = atomic_add(exec_cuda, &nPE[0], 1);
                  PE[no] = pair3_t{vi, tri[1], tri[2]};
                  csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] =
                      pair4_t{vi, tri[0], tri[1], tri[2]};
                  if (record) {
                    vtemp.template pack<3>("inds_pre", no) =
                        dpair3_t{vi, tri[1], tri[2]}.reinterpret_bits<T>();
                    vtemp("dist2_pre", no) = d2;
                  }
                }
                break;
              }
              case 5: {
                if (auto d2 = dist2_pe(p, t2, t0); d2 < dHat2) {
                  auto no = atomic_add(exec_cuda, &nPE[0], 1);
                  PE[no] = pair3_t{vi, tri[2], tri[0]};
                  csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] =
                      pair4_t{vi, tri[0], tri[1], tri[2]};
                  if (record) {
                    vtemp.template pack<3>("inds_pre", no) =
                        dpair3_t{vi, tri[2], tri[0]}.reinterpret_bits<T>();
                    vtemp("dist2_pre", no) = d2;
                  }
                }
                break;
              }
              case 6: {
                if (auto d2 = dist2_pt(p, t0, t1, t2); d2 < dHat2) {
                  auto no = atomic_add(exec_cuda, &nPT[0], 1);
                  PT[no] = pair4_t{vi, tri[0], tri[1], tri[2]};
                  csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] =
                      pair4_t{vi, tri[0], tri[1], tri[2]};
                  if (record) {
                    vtemp.template pack<4>("inds_pre", no) =
                        dpair4_t{vi, tri[0], tri[1], tri[2]}
                            .reinterpret_bits<T>();
                    vtemp("dist2_pre", no) = d2;
                  }
                }
                break;
              }
              default:
                break;
              }
            });
          });
      /// ee
      pol(Collapse{seInds.size()}, [seInds = proxy<space>({}, seInds),
                                    sedges = proxy<space>(
                                        {}, withBoundary ? coEdges : seInds),
                                    vtemp = proxy<space>({}, vtemp),
                                    bvh = proxy<space>(withBoundary ? bouSeBvh
                                                                    : seBvh),
                                    PP = proxy<space>(PP),
                                    nPP = proxy<space>(nPP),
                                    PE = proxy<space>(PE),
                                    nPE = proxy<space>(nPE),
                                    EE = proxy<space>(EE),
                                    nEE = proxy<space>(nEE),
                                    csEE = proxy<space>(csEE),
                                    ncsEE = proxy<space>(ncsEE), dHat, xi,
                                    thickness = xi + dHat,
                                    voffset = withBoundary ? coOffset : 0,
                                    record] __device__(int sei) mutable {
        const auto dHat2 = zs::sqr(dHat + xi);
        auto eiInds = seInds.template pack<2>("inds", sei)
                          .template reinterpret_bits<int>();
        bool selfFixed = vtemp("BCorder", eiInds[0]) == 3 &&
                         vtemp("BCorder", eiInds[1]) == 3;
        auto v0 = vtemp.template pack<3>("xn", eiInds[0]);
        auto v1 = vtemp.template pack<3>("xn", eiInds[1]);
        auto rv0 = vtemp.template pack<3>("x0", eiInds[0]);
        auto rv1 = vtemp.template pack<3>("x0", eiInds[1]);
        auto [mi, ma] = get_bounding_box(v0, v1);
        auto bv = bv_t{mi - thickness, ma + thickness};
        bvh.iter_neighbors(bv, [&](int sej) {
          if (voffset == 0 && sei < sej)
            return;
          auto ejInds = sedges.template pack<2>("inds", sej)
                            .template reinterpret_bits<int>() +
                        voffset;
          if (eiInds[0] == ejInds[0] || eiInds[0] == ejInds[1] ||
              eiInds[1] == ejInds[0] || eiInds[1] == ejInds[1])
            return;
          // all affected by sticky boundary conditions
          if (selfFixed && vtemp("BCorder", ejInds[0]) == 3 &&
              vtemp("BCorder", ejInds[1]) == 3)
            return;
          // ccd
          auto v2 = vtemp.template pack<3>("xn", ejInds[0]);
          auto v3 = vtemp.template pack<3>("xn", ejInds[1]);
          auto rv2 = vtemp.template pack<3>("x0", ejInds[0]);
          auto rv3 = vtemp.template pack<3>("x0", ejInds[1]);

          switch (ee_distance_type(v0, v1, v2, v3)) {
          case 0: {
            if (auto d2 = dist2_pp(v0, v2); d2 < dHat2) {
              {
                auto no = atomic_add(exec_cuda, &nPP[0], 1);
                PP[no] = pair_t{eiInds[0], ejInds[0]};
                csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] =
                    pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                if (record) {
                  vtemp.template pack<2>("inds_pre", no) =
                      dpair_t{eiInds[0], ejInds[0]}.reinterpret_bits<T>();
                  vtemp("dist2_pre", no) = d2;
                }
              }
            }
            break;
          }
          case 1: {
            if (auto d2 = dist2_pp(v0, v3); d2 < dHat2) {
              {
                auto no = atomic_add(exec_cuda, &nPP[0], 1);
                PP[no] = pair_t{eiInds[0], ejInds[1]};
                csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] =
                    pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                if (record) {
                  vtemp.template pack<2>("inds_pre", no) =
                      dpair_t{eiInds[0], ejInds[1]}.reinterpret_bits<T>();
                  vtemp("dist2_pre", no) = d2;
                }
              }
            }
            break;
          }
          case 2: {
            if (auto d2 = dist2_pe(v0, v2, v3); d2 < dHat2) {
              {
                auto no = atomic_add(exec_cuda, &nPE[0], 1);
                PE[no] = pair3_t{eiInds[0], ejInds[0], ejInds[1]};
                csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] =
                    pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                if (record) {
                  vtemp.template pack<3>("inds_pre", no) =
                      dpair3_t{eiInds[0], ejInds[0], ejInds[1]}
                          .reinterpret_bits<T>();
                  vtemp("dist2_pre", no) = d2;
                }
              }
            }
            break;
          }
          case 3: {
            if (auto d2 = dist2_pp(v1, v2); d2 < dHat2) {
              {
                auto no = atomic_add(exec_cuda, &nPP[0], 1);
                PP[no] = pair_t{eiInds[1], ejInds[0]};
                csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] =
                    pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                if (record) {
                  vtemp.template pack<2>("inds_pre", no) =
                      dpair_t{eiInds[1], ejInds[0]}.reinterpret_bits<T>();
                  vtemp("dist2_pre", no) = d2;
                }
              }
            }
            break;
          }
          case 4: {
            if (auto d2 = dist2_pp(v1, v3); d2 < dHat2) {
              {
                auto no = atomic_add(exec_cuda, &nPP[0], 1);
                PP[no] = pair_t{eiInds[1], ejInds[1]};
                csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] =
                    pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                if (record) {
                  vtemp.template pack<2>("inds_pre", no) =
                      dpair_t{eiInds[1], ejInds[1]}.reinterpret_bits<T>();
                  vtemp("dist2_pre", no) = d2;
                }
              }
            }
            break;
          }
          case 5: {
            if (auto d2 = dist2_pe(v1, v2, v3); d2 < dHat2) {
              {
                auto no = atomic_add(exec_cuda, &nPE[0], 1);
                PE[no] = pair3_t{eiInds[1], ejInds[0], ejInds[1]};
                csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] =
                    pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                if (record) {
                  vtemp.template pack<3>("inds_pre", no) =
                      dpair3_t{eiInds[1], ejInds[0], ejInds[1]}
                          .reinterpret_bits<T>();
                  vtemp("dist2_pre", no) = d2;
                }
              }
            }
            break;
          }
          case 6: {
            if (auto d2 = dist2_pe(v2, v0, v1); d2 < dHat2) {
              {
                auto no = atomic_add(exec_cuda, &nPE[0], 1);
                PE[no] = pair3_t{ejInds[0], eiInds[0], eiInds[1]};
                csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] =
                    pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                if (record) {
                  vtemp.template pack<3>("inds_pre", no) =
                      dpair3_t{ejInds[0], eiInds[0], eiInds[1]}
                          .reinterpret_bits<T>();
                  vtemp("dist2_pre", no) = d2;
                }
              }
            }
            break;
          }
          case 7: {
            if (auto d2 = dist2_pe(v3, v0, v1); d2 < dHat2) {
              {
                auto no = atomic_add(exec_cuda, &nPE[0], 1);
                PE[no] = pair3_t{ejInds[1], eiInds[0], eiInds[1]};
                csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] =
                    pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                if (record) {
                  vtemp.template pack<3>("inds_pre", no) =
                      dpair3_t{ejInds[1], eiInds[0], eiInds[1]}
                          .reinterpret_bits<T>();
                  vtemp("dist2_pre", no) = d2;
                }
              }
            }
            break;
          }
          case 8: {
            if (auto d2 = dist2_ee(v0, v1, v2, v3); d2 < dHat2) {
              {
                auto no = atomic_add(exec_cuda, &nEE[0], 1);
                EE[no] = pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] =
                    pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
                if (record) {
                  vtemp.template pack<4>("inds_pre", no) =
                      dpair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]}
                          .reinterpret_bits<T>();
                  vtemp("dist2_pre", no) = d2;
                }
              }
            }
            break;
          }
          default:
            break;
          }
        });
      });
    }
    bool checkSelfIntersection(zs::CudaExecutionPolicy &pol) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      const auto dHat2 = dHat * dHat;
      zs::Vector<int> intersected{vtemp.get_allocator(), 1};
      intersected.setVal(0);
      // self
      {
        auto edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", seInds,
                                                 zs::wrapv<2>{}, 0);
        bvh_t seBvh;
        seBvh.build(pol, edgeBvs);
        pol(Collapse{stInds.size()},
            [stInds = proxy<space>({}, stInds),
             seInds = proxy<space>({}, seInds), vtemp = proxy<space>({}, vtemp),
             intersected = proxy<space>(intersected),
             bvh = proxy<space>(seBvh)] __device__(int sti) mutable {
              auto tri = stInds.template pack<3>("inds", sti)
                             .template reinterpret_bits<int>();
              auto t0 = vtemp.pack<3>("xn", tri[0]);
              auto t1 = vtemp.pack<3>("xn", tri[1]);
              auto t2 = vtemp.pack<3>("xn", tri[2]);
              auto bv = bv_t{get_bounding_box(t0, t1)};
              merge(bv, t2);
              bool allFixed = vtemp("BCorder", tri[0]) == 3 &&
                              vtemp("BCorder", tri[1]) == 3 &&
                              vtemp("BCorder", tri[2]) == 3;
              bvh.iter_neighbors(bv, [&](int sei) {
                auto line = seInds.template pack<2>("inds", sei)
                                .template reinterpret_bits<int>();
                if (tri[0] == line[0] || tri[0] == line[1] ||
                    tri[1] == line[0] || tri[1] == line[1] ||
                    tri[2] == line[0] || tri[2] == line[1])
                  return;
                // ignore intersection under sticky boundary conditions
                if (allFixed && vtemp("BCorder", line[0]) == 3 &&
                    vtemp("BCorder", line[1]) == 3)
                  return;
                // ccd
                if (et_intersected(vtemp.pack<3>("xn", line[0]),
                                   vtemp.pack<3>("xn", line[1]), t0, t1, t2))
                  if (intersected[0] == 0)
                    intersected[0] = 1;
                // atomic_cas(exec_cuda, &intersected[0], 0, 1);
              });
            });
      }
      // boundary
      {
        auto edgeBvs = retrieve_bounding_volumes(pol, vtemp, "xn", coEdges,
                                                 zs::wrapv<2>{}, coOffset);
        bvh_t seBvh;
        seBvh.build(pol, edgeBvs);
        pol(Collapse{stInds.size()},
            [stInds = proxy<space>({}, stInds),
             coEdges = proxy<space>({}, coEdges),
             vtemp = proxy<space>({}, vtemp),
             intersected = proxy<space>(intersected), bvh = proxy<space>(seBvh),
             coOffset = coOffset] __device__(int sti) mutable {
              auto tri = stInds.template pack<3>("inds", sti)
                             .template reinterpret_bits<int>();
              auto t0 = vtemp.pack<3>("xn", tri[0]);
              auto t1 = vtemp.pack<3>("xn", tri[1]);
              auto t2 = vtemp.pack<3>("xn", tri[2]);
              auto bv = bv_t{get_bounding_box(t0, t1)};
              merge(bv, t2);
              bool allFixed = vtemp("BCorder", tri[0]) == 3 &&
                              vtemp("BCorder", tri[1]) == 3 &&
                              vtemp("BCorder", tri[2]) == 3;
              bvh.iter_neighbors(bv, [&](int sei) {
                auto line = coEdges.template pack<2>("inds", sei)
                                .template reinterpret_bits<int>() +
                            coOffset;
                // ignore intersection under sticky boundary conditions
                if (allFixed && vtemp("BCorder", line[0]) == 3 &&
                    vtemp("BCorder", line[1]) == 3)
                  return;
                // ccd
                if (et_intersected(vtemp.pack<3>("xn", line[0]),
                                   vtemp.pack<3>("xn", line[1]), t0, t1, t2))
                  if (intersected[0] == 0)
                    intersected[0] = 1;
                // atomic_cas(exec_cuda, &intersected[0], 0, 1);
              });
            });
      }
      return intersected.getVal();
    }
    void findCCDConstraints(zs::CudaExecutionPolicy &pol, T alpha, T xi = 0) {
      ncsPT.setVal(0);
      ncsEE.setVal(0);
      {
        auto triBvs = retrieve_bounding_volumes(
            pol, vtemp, "xn", stInds, zs::wrapv<3>{}, vtemp, "dir", alpha, 0);
        stBvh.refit(pol, triBvs);
        auto edgeBvs = retrieve_bounding_volumes(
            pol, vtemp, "xn", seInds, zs::wrapv<2>{}, vtemp, "dir", alpha, 0);
        seBvh.refit(pol, edgeBvs);
      }
      findCCDConstraintsImpl(pol, alpha, xi, false);

      {
        auto triBvs =
            retrieve_bounding_volumes(pol, vtemp, "xn", coEles, zs::wrapv<3>{},
                                      vtemp, "dir", alpha, coOffset);
        bouStBvh.refit(pol, triBvs);
        auto edgeBvs =
            retrieve_bounding_volumes(pol, vtemp, "xn", coEdges, zs::wrapv<2>{},
                                      vtemp, "dir", alpha, coOffset);
        bouSeBvh.refit(pol, edgeBvs);
      }
      findCCDConstraintsImpl(pol, alpha, xi, true);
    }
    void findCCDConstraintsImpl(zs::CudaExecutionPolicy &pol, T alpha, T xi,
                                bool withBoundary = false) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      const auto dHat2 = dHat * dHat;

      /// pt
      pol(Collapse{svInds.size()},
          [svInds = proxy<space>({}, svInds),
           eles = proxy<space>({}, withBoundary ? coEles : stInds),
           vtemp = proxy<space>({}, vtemp),
           bvh = proxy<space>(withBoundary ? bouStBvh : stBvh),
           PP = proxy<space>(PP), nPP = proxy<space>(nPP),
           PE = proxy<space>(PE), nPE = proxy<space>(nPE),
           PT = proxy<space>(PT), nPT = proxy<space>(nPT),
           csPT = proxy<space>(csPT), ncsPT = proxy<space>(ncsPT), xi, alpha,
           voffset = withBoundary ? coOffset : 0] __device__(int vi) mutable {
            vi = reinterpret_bits<int>(svInds("inds", vi));
            auto p = vtemp.template pack<3>("xn", vi);
            auto dir = vtemp.template pack<3>("dir", vi);
            auto bv = bv_t{get_bounding_box(p, p + alpha * dir)};
            bv._min -= xi;
            bv._max += xi;
            bvh.iter_neighbors(bv, [&](int stI) {
              auto tri = eles.template pack<3>("inds", stI)
                             .template reinterpret_bits<int>() +
                         voffset;
              if (vi == tri[0] || vi == tri[1] || vi == tri[2])
                return;
              // all affected by sticky boundary conditions
              if (vtemp("BCorder", vi) == 3 && vtemp("BCorder", tri[0]) == 3 &&
                  vtemp("BCorder", tri[1]) == 3 &&
                  vtemp("BCorder", tri[2]) == 3)
                return;
              csPT[atomic_add(exec_cuda, &ncsPT[0], 1)] =
                  pair4_t{vi, tri[0], tri[1], tri[2]};
            });
          });
      /// ee
      pol(Collapse{seInds.size()},
          [seInds = proxy<space>({}, seInds),
           sedges = proxy<space>({}, withBoundary ? coEdges : seInds),
           vtemp = proxy<space>({}, vtemp),
           bvh = proxy<space>(withBoundary ? bouSeBvh : seBvh),
           PP = proxy<space>(PP), nPP = proxy<space>(nPP),
           PE = proxy<space>(PE), nPE = proxy<space>(nPE),
           EE = proxy<space>(PT), nEE = proxy<space>(nPT),
           csEE = proxy<space>(csEE), ncsEE = proxy<space>(ncsEE), xi, alpha,
           voffset = withBoundary ? coOffset : 0] __device__(int sei) mutable {
            auto eiInds = seInds.template pack<2>("inds", sei)
                              .template reinterpret_bits<int>();
            bool selfFixed = vtemp("BCorder", eiInds[0]) == 3 &&
                             vtemp("BCorder", eiInds[1]) == 3;
            auto v0 = vtemp.template pack<3>("xn", eiInds[0]);
            auto v1 = vtemp.template pack<3>("xn", eiInds[1]);
            auto dir0 = vtemp.template pack<3>("dir", eiInds[0]);
            auto dir1 = vtemp.template pack<3>("dir", eiInds[1]);
            auto bv = bv_t{get_bounding_box(v0, v0 + alpha * dir0)};
            merge(bv, v1);
            merge(bv, v1 + alpha * dir1);
            bv._min -= xi;
            bv._max += xi;
            bvh.iter_neighbors(bv, [&](int sej) {
              if (voffset == 0 && sei < sej)
                return;
              auto ejInds = sedges.template pack<2>("inds", sej)
                                .template reinterpret_bits<int>() +
                            voffset;
              if (eiInds[0] == ejInds[0] || eiInds[0] == ejInds[1] ||
                  eiInds[1] == ejInds[0] || eiInds[1] == ejInds[1])
                return;
              // all affected by sticky boundary conditions
              if (selfFixed && vtemp("BCorder", ejInds[0]) == 3 &&
                  vtemp("BCorder", ejInds[1]) == 3)
                return;
              csEE[atomic_add(exec_cuda, &ncsEE[0], 1)] =
                  pair4_t{eiInds[0], eiInds[1], ejInds[0], ejInds[1]};
            });
          });
    }
    ///
    void computeBarrierGradientAndHessian(zs::CudaExecutionPolicy &pol,
                                          const zs::SmallString &gTag = "grad",
                                          bool includeHessian = true);

    void intersectionFreeStepsize(zs::CudaExecutionPolicy &pol, T xi,
                                  T &stepSize) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;

      Vector<T> alpha{vtemp.get_allocator(), 1};
      alpha.setVal(stepSize);
      auto npt = ncsPT.getVal();
      pol(range(npt),
          [csPT = proxy<space>(csPT), vtemp = proxy<space>({}, vtemp),
           alpha = proxy<space>(alpha), stepSize, xi,
           coOffset = (int)coOffset] __device__(int pti) {
            auto ids = csPT[pti];
            auto p = vtemp.template pack<3>("xn", ids[0]);
            auto t0 = vtemp.template pack<3>("xn", ids[1]);
            auto t1 = vtemp.template pack<3>("xn", ids[2]);
            auto t2 = vtemp.template pack<3>("xn", ids[3]);
            auto dp = vtemp.template pack<3>("dir", ids[0]);
            auto dt0 = vtemp.template pack<3>("dir", ids[1]);
            auto dt1 = vtemp.template pack<3>("dir", ids[2]);
            auto dt2 = vtemp.template pack<3>("dir", ids[3]);
            T tmp = alpha[0];
#if 1
            if (accd::ptccd(p, t0, t1, t2, dp, dt0, dt1, dt2, (T)0.2, xi, tmp))
#elif 1
            if (ticcd::ptccd(p, t0, t1, t2, dp, dt0, dt1, dt2, (T)0.2, xi, tmp))
#else
            if (pt_ccd(p, t0, t1, t2, dp, dt0, dt1, dt2, xi, tmp))
#endif
              atomic_min(exec_cuda, &alpha[0], tmp);
          });
      auto nee = ncsEE.getVal();
      pol(range(nee),
          [csEE = proxy<space>(csEE), vtemp = proxy<space>({}, vtemp),
           alpha = proxy<space>(alpha), stepSize, xi,
           coOffset = (int)coOffset] __device__(int eei) {
            auto ids = csEE[eei];
            auto ea0 = vtemp.template pack<3>("xn", ids[0]);
            auto ea1 = vtemp.template pack<3>("xn", ids[1]);
            auto eb0 = vtemp.template pack<3>("xn", ids[2]);
            auto eb1 = vtemp.template pack<3>("xn", ids[3]);
            auto dea0 = vtemp.template pack<3>("dir", ids[0]);
            auto dea1 = vtemp.template pack<3>("dir", ids[1]);
            auto deb0 = vtemp.template pack<3>("dir", ids[2]);
            auto deb1 = vtemp.template pack<3>("dir", ids[3]);
            auto tmp = alpha[0];
#if 1
            if (accd::eeccd(ea0, ea1, eb0, eb1, dea0, dea1, deb0, deb1, (T)0.2,
                            xi, tmp))
#elif 1
            if (ticcd::eeccd(ea0, ea1, eb0, eb1, dea0, dea1, deb0, deb1, (T)0.2, xi, tmp))
#else
            if (ee_ccd(ea0, ea1, eb0, eb1, dea0, dea1, deb0, deb1, xi, tmp))
#endif
              atomic_min(exec_cuda, &alpha[0], tmp);
          });
      stepSize = alpha.getVal();
    }
    void groundIntersectionFreeStepsize(zs::CudaExecutionPolicy &pol,
                                        T &stepSize) {
      using namespace zs;
      // constexpr T slackness = 0.8;
      constexpr auto space = execspace_e::cuda;

      zs::Vector<T> finalAlpha{coVerts.get_allocator(), 1};
      finalAlpha.setVal(stepSize);
      pol(Collapse{coOffset},
          [vtemp = proxy<space>({}, vtemp),
           // boundary
           gn = s_groundNormal, finalAlpha = proxy<space>(finalAlpha),
           stepSize] ZS_LAMBDA(int vi) mutable {
            // this vert affected by sticky boundary conditions
            if (vtemp("BCorder", vi) == 3)
              return;
            auto dir = vtemp.pack<3>("dir", vi);
            auto coef = gn.dot(dir);
            if (coef < 0) { // impacting direction
              auto x = vtemp.pack<3>("xn", vi);
              auto dist = gn.dot(x);
              auto maxAlpha = (dist * 0.8) / (-coef);
              if (maxAlpha < stepSize)
                atomic_min(exec_cuda, &finalAlpha[0], maxAlpha);
            }
          });
      stepSize = finalAlpha.getVal();
      fmt::print(fg(fmt::color::dark_cyan), "ground alpha: {}\n", stepSize);
    }
    ///
    void computeBoundaryBarrierGradientAndHessian(
        zs::CudaExecutionPolicy &pol, const zs::SmallString &gTag = "grad",
        bool includeHessian = true) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      pol(range(coOffset),
          [vtemp = proxy<space>({}, vtemp), tempPB = proxy<space>({}, tempPB),
           gTag, gn = s_groundNormal, dHat2 = dHat * dHat, kappa = kappa,
           projectDBC = projectDBC, includeHessian] ZS_LAMBDA(int vi) mutable {
            auto x = vtemp.pack<3>("xn", vi);
            auto dist = gn.dot(x);
            auto dist2 = dist * dist;
            auto t = dist2 - dHat2;
            auto g_b = t * zs::log(dist2 / dHat2) * -2 - (t * t) / dist2;
            auto H_b = (zs::log(dist2 / dHat2) * -2.0 - t * 4.0 / dist2) +
                       1.0 / (dist2 * dist2) * (t * t);
            if (dist2 < dHat2) {
              auto grad = -gn * (kappa * g_b * 2 * dist);
              for (int d = 0; d != 3; ++d)
                atomic_add(exec_cuda, &vtemp(gTag, d, vi), grad(d));
            }

            if (!includeHessian)
              return;
            auto param = 4 * H_b * dist2 + 2 * g_b;
            auto hess = mat3::zeros();
            if (dist2 < dHat2 && param > 0) {
              auto nn = dyadic_prod(gn, gn);
              hess = (kappa * param) * nn;
            }

            // make_pd(hess);
            mat3 BCbasis[1] = {vtemp.pack<3, 3>("BCbasis", vi)};
            int BCorder[1] = {(int)vtemp("BCorder", vi)};
            int BCfixed[1] = {(int)vtemp("BCfixed", vi)};
            rotate_hessian(hess, BCbasis, BCorder, BCfixed, projectDBC);
            tempPB.tuple<9>("H", vi) = hess;
            for (int i = 0; i != 3; ++i)
              for (int j = 0; j != 3; ++j) {
                atomic_add(exec_cuda, &vtemp("P", i * 3 + j, vi), hess(i, j));
              }
          });
      return;
    }
    template <typename Model>
    void computeElasticGradientAndHessian(zs::CudaExecutionPolicy &cudaPol,
                                          const Model &model,
                                          const zs::SmallString &gTag = "grad",
                                          bool includeHessian = true) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      for (auto &primHandle : prims)
        if (primHandle.category == ZenoParticles::surface)
          cudaPol(
              zs::range(primHandle.getEles().size()),
              [vtemp = proxy<space>({}, vtemp),
               etemp = proxy<space>({}, primHandle.etemp),
               eles = proxy<space>({}, primHandle.getEles()), model, gTag,
               dt = this->dt, projectDBC = projectDBC,
               vOffset = primHandle.vOffset,
               includeHessian] __device__(int ei) mutable {
                auto IB = eles.template pack<2, 2>("IB", ei);
                auto inds = eles.template pack<3>("inds", ei)
                                .template reinterpret_bits<int>() +
                            vOffset;
                auto vole = eles("vol", ei);
                vec3 xs[3] = {vtemp.template pack<3>("xn", inds[0]),
                              vtemp.template pack<3>("xn", inds[1]),
                              vtemp.template pack<3>("xn", inds[2])};
                auto x1x0 = xs[1] - xs[0];
                auto x2x0 = xs[2] - xs[0];

                mat3 BCbasis[3];
                int BCorder[3];
                int BCfixed[3];
                for (int i = 0; i != 3; ++i) {
                  BCbasis[i] = vtemp.pack<3, 3>("BCbasis", inds[i]);
                  BCorder[i] = vtemp("BCorder", inds[i]);
                  BCfixed[i] = vtemp("BCfixed", inds[i]);
                }
                zs::vec<T, 9, 9> H;
                if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3) {
                  etemp.tuple<9 * 9>("He", ei) = H.zeros();
                  return;
                }

                zs::vec<T, 3, 2> Ds{x1x0[0], x2x0[0], x1x0[1],
                                    x2x0[1], x1x0[2], x2x0[2]};
                auto F = Ds * IB;

                auto dFdX = dFdXMatrix(IB, wrapv<3>{});
                auto dFdXT = dFdX.transpose();
                auto f0 = col(F, 0);
                auto f1 = col(F, 1);
                auto f0Norm = zs::sqrt(f0.l2NormSqr());
                auto f1Norm = zs::sqrt(f1.l2NormSqr());
                auto f0Tf1 = f0.dot(f1);
                zs::vec<T, 3, 2> Pstretch, Pshear;
                for (int d = 0; d != 3; ++d) {
                  Pstretch(d, 0) = 2 * (1 - 1 / f0Norm) * F(d, 0);
                  Pstretch(d, 1) = 2 * (1 - 1 / f1Norm) * F(d, 1);
                  Pshear(d, 0) = 2 * f0Tf1 * f1(d);
                  Pshear(d, 1) = 2 * f0Tf1 * f0(d);
                }
                auto vecP =
                    flatten(model.mu * Pstretch + (model.mu * 0.3) * Pshear);
                auto vfdt2 = -vole * (dFdXT * vecP) * (dt * dt);

                for (int i = 0; i != 3; ++i) {
                  auto vi = inds[i];
                  for (int d = 0; d != 3; ++d)
                    atomic_add(exec_cuda, &vtemp(gTag, d, vi),
                               (T)vfdt2(i * 3 + d));
                }

                if (!includeHessian)
                  return;
                /// ref: A Finite Element Formulation of Baraff-Witkin Cloth
                // suggested by huang kemeng
                auto stretchHessian = [&F, &model]() {
                  auto H = zs::vec<T, 6, 6>::zeros();
                  const zs::vec<T, 2> u{1, 0};
                  const zs::vec<T, 2> v{0, 1};
                  const T I5u = (F * u).l2NormSqr();
                  const T I5v = (F * v).l2NormSqr();
                  const T invSqrtI5u = (T)1 / zs::sqrt(I5u);
                  const T invSqrtI5v = (T)1 / zs::sqrt(I5v);

                  H(0, 0) = H(1, 1) = H(2, 2) = zs::max(1 - invSqrtI5u, (T)0);
                  H(3, 3) = H(4, 4) = H(5, 5) = zs::max(1 - invSqrtI5v, (T)0);

                  const auto fu = col(F, 0).normalized();
                  const T uCoeff = (1 - invSqrtI5u >= 0) ? invSqrtI5u : (T)1;
                  for (int i = 0; i != 3; ++i)
                    for (int j = 0; j != 3; ++j)
                      H(i, j) += uCoeff * fu(i) * fu(j);

                  const auto fv = col(F, 1).normalized();
                  const T vCoeff = (1 - invSqrtI5v >= 0) ? invSqrtI5v : (T)1;
                  for (int i = 0; i != 3; ++i)
                    for (int j = 0; j != 3; ++j)
                      H(3 + i, 3 + j) += vCoeff * fv(i) * fv(j);

                  H *= model.mu;
                  return H;
                };
                auto shearHessian = [&F, &model]() {
                  using mat6 = zs::vec<T, 6, 6>;
                  auto H = mat6::zeros();
                  const zs::vec<T, 2> u{1, 0};
                  const zs::vec<T, 2> v{0, 1};
                  const T I6 = (F * u).dot(F * v);
                  const T signI6 = I6 >= 0 ? 1 : -1;

                  H(3, 0) = H(4, 1) = H(5, 2) = H(0, 3) = H(1, 4) = H(2, 5) =
                      (T)1;

                  const auto g_ = F * (dyadic_prod(u, v) + dyadic_prod(v, u));
                  zs::vec<T, 6> g{};
                  for (int j = 0, offset = 0; j != 2; ++j) {
                    for (int i = 0; i != 3; ++i)
                      g(offset++) = g_(i, j);
                  }

                  const T I2 = F.l2NormSqr();
                  const T lambda0 =
                      (T)0.5 * (I2 + zs::sqrt(I2 * I2 + (T)12 * I6 * I6));

                  const zs::vec<T, 6> q0 =
                      (I6 * H * g + lambda0 * g).normalized();

                  auto t = mat6::identity();
                  t = 0.5 * (t + signI6 * H);

                  const zs::vec<T, 6> Tq = t * q0;
                  const auto normTq = Tq.l2NormSqr();

                  mat6 dPdF =
                      zs::abs(I6) * (t - (dyadic_prod(Tq, Tq) / normTq)) +
                      lambda0 * (dyadic_prod(q0, q0));
                  dPdF *= (model.mu * 0.3);
                  return dPdF;
                };
                auto He = stretchHessian() + shearHessian();
                H = dFdX.transpose() * He * dFdX;
                H *= dt * dt * vole;

                // rotate and project
                rotate_hessian(H, BCbasis, BCorder, BCfixed, projectDBC);
                etemp.tuple<9 * 9>("He", ei) = H;
                for (int vi = 0; vi != 3; ++vi) {
                  for (int i = 0; i != 3; ++i)
                    for (int j = 0; j != 3; ++j) {
                      atomic_add(exec_cuda, &vtemp("P", i * 3 + j, inds[vi]),
                                 H(vi * 3 + i, vi * 3 + j));
                    }
                }
              });
        else if (primHandle.category == ZenoParticles::tet)
          cudaPol(
              zs::range(primHandle.getEles().size()),
              [vtemp = proxy<space>({}, vtemp),
               etemp = proxy<space>({}, primHandle.etemp),
               eles = proxy<space>({}, primHandle.getEles()), model, gTag,
               dt = this->dt, projectDBC = projectDBC,
               vOffset = primHandle.vOffset,
               includeHessian] __device__(int ei) mutable {
                auto IB = eles.template pack<3, 3>("IB", ei);
                auto inds = eles.template pack<4>("inds", ei)
                                .template reinterpret_bits<int>() +
                            vOffset;
                auto vole = eles("vol", ei);
                vec3 xs[4] = {
                    vtemp.pack<3>("xn", inds[0]), vtemp.pack<3>("xn", inds[1]),
                    vtemp.pack<3>("xn", inds[2]), vtemp.pack<3>("xn", inds[3])};

                mat3 BCbasis[4];
                int BCorder[4];
                int BCfixed[4];
                for (int i = 0; i != 4; ++i) {
                  BCbasis[i] = vtemp.pack<3, 3>("BCbasis", inds[i]);
                  BCorder[i] = vtemp("BCorder", inds[i]);
                  BCfixed[i] = vtemp("BCfixed", inds[i]);
                }
                zs::vec<T, 12, 12> H;
                if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3 &&
                    BCorder[3] == 3) {
                  etemp.tuple<12 * 12>("He", ei) = H.zeros();
                  return;
                }
                mat3 F{};
                {
                  auto x1x0 = xs[1] - xs[0];
                  auto x2x0 = xs[2] - xs[0];
                  auto x3x0 = xs[3] - xs[0];
                  auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                 x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                  F = Ds * IB;
                }
                auto P = model.first_piola(F);
                auto vecP = flatten(P);
                auto dFdX = dFdXMatrix(IB);
                auto dFdXT = dFdX.transpose();
                auto vfdt2 = -vole * (dFdXT * vecP) * dt * dt;

                for (int i = 0; i != 4; ++i) {
                  auto vi = inds[i];
                  for (int d = 0; d != 3; ++d)
                    atomic_add(exec_cuda, &vtemp(gTag, d, vi),
                               (T)vfdt2(i * 3 + d));
                }

                if (!includeHessian)
                  return;
                auto Hq = model.first_piola_derivative(F, true_c);
                H = dFdXT * Hq * dFdX * vole * dt * dt;

                // rotate and project
                rotate_hessian(H, BCbasis, BCorder, BCfixed, projectDBC);
                etemp.tuple<12 * 12>("He", ei) = H;
                for (int vi = 0; vi != 4; ++vi) {
                  for (int i = 0; i != 3; ++i)
                    for (int j = 0; j != 3; ++j) {
                      atomic_add(exec_cuda, &vtemp("P", i * 3 + j, inds[vi]),
                                 H(vi * 3 + i, vi * 3 + j));
                    }
                }
              });
    }
    void computeInertialAndGravityPotentialGradient(
        zs::CudaExecutionPolicy &cudaPol,
        const zs::SmallString &gTag = "grad") {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      cudaPol(zs::range(coOffset),
              [tempPB = proxy<space>({}, tempPB),
               vtemp = proxy<space>({}, vtemp), gTag, extForce = extForce,
               dt = dt, projectDBC = projectDBC] __device__(int i) mutable {
                auto m = zs::sqr(vtemp("ws", i));
                vtemp.tuple<3>(gTag, i) =
                    m * extForce * dt * dt -
                    m * (vtemp.pack<3>("xn", i) - vtemp.pack<3>("xtilde", i));

                auto M = mat3::identity() * m;
                mat3 BCbasis[1] = {vtemp.template pack<3, 3>("BCbasis", i)};
                int BCorder[1] = {(int)vtemp("BCorder", i)};
                int BCfixed[1] = {(int)vtemp("BCfixed", i)};
                rotate_hessian(M, BCbasis, BCorder, BCfixed, projectDBC);
                tempPB.template tuple<9>("Hi", i) = M;
                // prepare preconditioner
                for (int r = 0; r != 3; ++r)
                  for (int c = 0; c != 3; ++c)
                    vtemp("P", r * 3 + c, i) += M(r, c);
              });
#if 0
              // collision object
      cudaPol(zs::range(coVerts.size()),
              [vtemp = proxy<space>({}, vtemp),
               coverts = proxy<space>({}, coVerts), projectDBC = projectDBC,
               gTag, coOffset = coOffset, dt = dt] __device__(int i) mutable {
                i += coOffset;
                auto m = zs::sqr(vtemp("ws", i));
                // only inertial, overwrite
                if (projectDBC || vtemp("BCfixed", i) == 1)
                  vtemp.tuple<3>(gTag, i) = vec3::zeros();
                else
                  vtemp.tuple<3>(gTag, i) = -m * (vtemp.pack<3>("xn", i) -
                                                  vtemp.pack<3>("xtilde", i));
                vtemp("P", 0, i) += m;
                vtemp("P", 4, i) += m;
                vtemp("P", 8, i) += m;
              });
#endif
    }
#if 1
    template <typename Model>
    T energy(zs::CudaExecutionPolicy &pol, const Model &model,
             const zs::SmallString tag, bool includeAugLagEnergy = false) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      Vector<T> &es = temp;

      es.resize(count_warps(coOffset));
      es.reset(0);
      std::vector<T> Es(0);

      // inertial
      pol(range(coOffset),
          [vtemp = proxy<space>({}, vtemp), es = proxy<space>(es), tag,
           extForce = extForce, dt = this->dt,
           n = coOffset] __device__(int vi) mutable {
            auto m = zs::sqr(vtemp("ws", vi));
            auto x = vtemp.pack<3>(tag, vi);
            int BCorder = vtemp("BCorder", vi);
            T E = 0;
            if (BCorder != 3) {
              // inertia
              E += (T)0.5 * m * (x - vtemp.pack<3>("xtilde", vi)).l2NormSqr();
              // external force
              E += -m * extForce.dot(x) * dt * dt;
            }
            reduce_to(vi, n, E, es[vi / 32]);
          });
      Es.push_back(reduce(pol, es));

      for (auto &primHandle : prims) {
        auto &verts = primHandle.getVerts();
        auto &eles = primHandle.getEles();
        es.resize(count_warps(eles.size()));
        es.reset(0);
        if (primHandle.category == ZenoParticles::surface) {
          // elasticity
          pol(range(eles.size()), [eles = proxy<space>({}, eles),
                                   vtemp = proxy<space>({}, vtemp),
                                   es = proxy<space>(es), tag, model = model,
                                   vOffset = primHandle.vOffset,
                                   n = eles.size()] __device__(int ei) mutable {
            auto IB = eles.template pack<2, 2>("IB", ei);
            auto inds = eles.template pack<3>("inds", ei)
                            .template reinterpret_bits<int>() +
                        vOffset;

            int BCorder[3];
            for (int i = 0; i != 3; ++i)
              BCorder[i] = vtemp("BCorder", inds[i]);
            T E;
            if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3)
              E = 0;
            else {
              auto vole = eles("vol", ei);
              vec3 xs[3] = {vtemp.template pack<3>(tag, inds[0]),
                            vtemp.template pack<3>(tag, inds[1]),
                            vtemp.template pack<3>(tag, inds[2])};
              auto x1x0 = xs[1] - xs[0];
              auto x2x0 = xs[2] - xs[0];

              zs::vec<T, 3, 2> Ds{x1x0[0], x2x0[0], x1x0[1],
                                  x2x0[1], x1x0[2], x2x0[2]};
              auto F = Ds * IB;
              auto f0 = col(F, 0);
              auto f1 = col(F, 1);
              auto f0Norm = zs::sqrt(f0.l2NormSqr());
              auto f1Norm = zs::sqrt(f1.l2NormSqr());
              auto Estretch =
                  model.mu * vole * (zs::sqr(f0Norm - 1) + zs::sqr(f1Norm - 1));
              auto Eshear = (model.mu * 0.3) * vole * zs::sqr(f0.dot(f1));
              E = Estretch + Eshear;
            }
            // atomic_add(exec_cuda, &res[0], E);
            // es[ei] = E;
            reduce_to(ei, n, E, es[ei / 32]);
          });
          Es.push_back(reduce(pol, es) * dt * dt);
        } else if (primHandle.category == ZenoParticles::tet) {
          pol(zs::range(eles.size()),
              [vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, eles),
               es = proxy<space>(es), model, tag, vOffset = primHandle.vOffset,
               n = eles.size()] __device__(int ei) mutable {
                auto IB = eles.template pack<3, 3>("IB", ei);
                auto inds = eles.template pack<4>("inds", ei)
                                .template reinterpret_bits<int>() +
                            vOffset;
                auto vole = eles("vol", ei);
                vec3 xs[4] = {
                    vtemp.pack<3>(tag, inds[0]), vtemp.pack<3>(tag, inds[1]),
                    vtemp.pack<3>(tag, inds[2]), vtemp.pack<3>(tag, inds[3])};

                int BCorder[4];
                for (int i = 0; i != 4; ++i)
                  BCorder[i] = vtemp("BCorder", inds[i]);
                T E;
                if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3 &&
                    BCorder[3] == 3)
                  E = 0;
                else {
                  mat3 F{};
                  auto x1x0 = xs[1] - xs[0];
                  auto x2x0 = xs[2] - xs[0];
                  auto x3x0 = xs[3] - xs[0];
                  auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                 x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                  F = Ds * IB;
                  E = model.psi(F) * vole;
                }
                // atomic_add(exec_cuda, &res[0], model.psi(F) * vole);
                // es[ei] = model.psi(F) * vole;
                reduce_to(ei, n, E, es[ei / 32]);
              });
          Es.push_back(reduce(pol, es) * dt * dt);
        }
      }
#if 0
      // collision object
      es.resize(count_warps(coVerts.size()));
      es.reset(0);
      // boundary inertial
      pol(range(coVerts.size()),
          [vtemp = proxy<space>({}, vtemp), es = proxy<space>(es), tag,
           projectDBC = projectDBC, coOffset = coOffset,
           n = coVerts.size()] __device__(int vi) mutable {
            vi += coOffset;
            // inertia
            auto m = zs::sqr(vtemp("ws", vi));
            auto x = vtemp.pack<3>(tag, vi);

            T E;
            if (projectDBC || vtemp("BCfixed", vi) == 1)
              E = 0;
            else
              E = (T)0.5 * m * (x - vtemp.pack<3>("xtilde", vi)).l2NormSqr();
            reduce_to(threadIdx.x, n, E, es[threadIdx.x / 32]);
          });
      Es.push_back(reduce(pol, es));
#endif
      // contacts
      {
#if s_enableContact
        {
          auto activeGap2 = dHat * dHat + 2 * xi * dHat;
          auto numPP = nPP.getVal();
          es.resize(count_warps(numPP));
          es.reset(0);
          pol(range(numPP),
              [vtemp = proxy<space>({}, vtemp),
               tempPP = proxy<space>({}, tempPP), PP = proxy<space>(PP),
               es = proxy<space>(es), xi2 = xi * xi, dHat = dHat, activeGap2,
               n = numPP] __device__(int ppi) mutable {
                auto pp = PP[ppi];
                auto x0 = vtemp.pack<3>("xn", pp[0]);
                auto x1 = vtemp.pack<3>("xn", pp[1]);
                auto dist2 = dist2_pp(x0, x1);
                if (dist2 < xi2)
                  printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                // es[ppi] = zs::barrier(dist2 - xi2, activeGap2, (T)1);
                reduce_to(ppi, n, zs::barrier(dist2 - xi2, activeGap2, (T)1),
                          es[ppi / 32]);
              });
          Es.push_back(reduce(pol, es) * kappa);

          auto numPE = nPE.getVal();
          es.resize(count_warps(numPE));
          es.reset(0);
          pol(range(numPE),
              [vtemp = proxy<space>({}, vtemp),
               tempPE = proxy<space>({}, tempPE), PE = proxy<space>(PE),
               es = proxy<space>(es), xi2 = xi * xi, dHat = dHat, activeGap2,
               n = numPE] __device__(int pei) mutable {
                auto pe = PE[pei];
                auto p = vtemp.pack<3>("xn", pe[0]);
                auto e0 = vtemp.pack<3>("xn", pe[1]);
                auto e1 = vtemp.pack<3>("xn", pe[2]);

                auto dist2 = dist2_pe(p, e0, e1);
                if (dist2 < xi2)
                  printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                // es[pei] = zs::barrier(dist2 - xi2, activeGap2, (T)1);
                reduce_to(pei, n, zs::barrier(dist2 - xi2, activeGap2, (T)1),
                          es[pei / 32]);
              });
          Es.push_back(reduce(pol, es) * kappa);

          auto numPT = nPT.getVal();
          es.resize(count_warps(numPT));
          es.reset(0);
          pol(range(numPT),
              [vtemp = proxy<space>({}, vtemp),
               tempPT = proxy<space>({}, tempPT), PT = proxy<space>(PT),
               es = proxy<space>(es), xi2 = xi * xi, dHat = dHat, activeGap2,
               n = numPT] __device__(int pti) mutable {
                auto pt = PT[pti];
                auto p = vtemp.pack<3>("xn", pt[0]);
                auto t0 = vtemp.pack<3>("xn", pt[1]);
                auto t1 = vtemp.pack<3>("xn", pt[2]);
                auto t2 = vtemp.pack<3>("xn", pt[3]);

                auto dist2 = dist2_pt(p, t0, t1, t2);
                if (dist2 < xi2)
                  printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                // es[pti] = zs::barrier(dist2 - xi2, activeGap2, (T)1);
                reduce_to(pti, n, zs::barrier(dist2 - xi2, activeGap2, (T)1),
                          es[pti / 32]);
              });
          Es.push_back(reduce(pol, es) * kappa);

          auto numEE = nEE.getVal();
          es.resize(count_warps(numEE));
          es.reset(0);
          pol(range(numEE),
              [vtemp = proxy<space>({}, vtemp),
               tempEE = proxy<space>({}, tempEE), EE = proxy<space>(EE),
               es = proxy<space>(es), xi2 = xi * xi, dHat = dHat, activeGap2,
               n = numEE] __device__(int eei) mutable {
                auto ee = EE[eei];
                auto ea0 = vtemp.pack<3>("xn", ee[0]);
                auto ea1 = vtemp.pack<3>("xn", ee[1]);
                auto eb0 = vtemp.pack<3>("xn", ee[2]);
                auto eb1 = vtemp.pack<3>("xn", ee[3]);

                auto dist2 = dist2_ee(ea0, ea1, eb0, eb1);
                if (dist2 < xi2)
                  printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                // es[eei] = zs::barrier(dist2 - xi2, activeGap2, (T)1);
                reduce_to(eei, n, zs::barrier(dist2 - xi2, activeGap2, (T)1),
                          es[eei / 32]);
              });
          Es.push_back(reduce(pol, es) * kappa);
        }
#endif
        // boundary
        es.resize(count_warps(coOffset));
        es.reset(0);
        pol(range(coOffset),
            [vtemp = proxy<space>({}, vtemp), es = proxy<space>(es),
             gn = s_groundNormal, dHat2 = dHat * dHat,
             n = coOffset] ZS_LAMBDA(int vi) mutable {
              auto x = vtemp.pack<3>("xn", vi);
              auto dist = gn.dot(x);
              auto dist2 = dist * dist;
              T E;
              if (dist2 < dHat2) {
                auto temp =
                    -(dist2 - dHat2) * (dist2 - dHat2) * zs::log(dist2 / dHat2);
                // atomic_add(exec_cuda, &res[0], temp);
                E = temp;
              } else
                E = 0;
              reduce_to(vi, n, E, es[vi / 32]);
            });
        Es.push_back(reduce(pol, es) * kappa);
      }
      // constraints
      if (includeAugLagEnergy) {
        computeConstraints(pol, tag);
        es.resize(count_warps(numDofs));
        es.reset(0);
        pol(range(numDofs),
            [vtemp = proxy<space>({}, vtemp), es = proxy<space>(es),
             n = numDofs] __device__(int vi) mutable {
              // already updated during "xn" update
              auto cons = vtemp.template pack<3>("cons", vi);
              auto w = vtemp("ws", vi);
              auto lambda = vtemp.pack<3>("lambda", vi);
#if 0
          atomic_add(
              exec_cuda, &res[0],
              (T)(-lambda.dot(cons) * w + 0.5 * w * cons.l2NormSqr()));
#else
          // es[vi] = (T)(-lambda.dot(cons) * w + 0.5 * w * cons.l2NormSqr());
              reduce_to(vi, n, (T)(-lambda.dot(cons) * w + 0.5 * w * cons.l2NormSqr()), es[vi / 32]);
#endif
            });
        Es.push_back(reduce(pol, es) * kappa);
      }
      std::sort(Es.begin(), Es.end());
      T E = 0;
      for (auto e : Es)
        E += e;
      return E;
    }
#else
    template <typename Model>
    T energy(zs::CudaExecutionPolicy &pol, const Model &model,
             const zs::SmallString tag, bool includeAugLagEnergy = false) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      Vector<T> res{vtemp.get_allocator(), 1};
      res.setVal(0);
      for (auto &primHandle : prims) {
        auto &verts = primHandle.getVerts();
        auto &eles = primHandle.getEles();
        pol(range(verts.size()),
            [verts = proxy<space>({}, verts), vtemp = proxy<space>({}, vtemp),
             res = proxy<space>(res), tag, extForce = extForce, dt = this->dt,
             vOffset = primHandle.vOffset,
             numVerts = verts.size()] __device__(int vi) mutable {
              // inertia
              auto m = verts("m", vi);
              vi += vOffset;
              auto x = vtemp.pack<3>(tag, vi);
              int BCorder = vtemp("BCorder", vi);
              T E = 0;
              if (BCorder != 3) {
                E += (T)0.5 * m * (x - vtemp.pack<3>("xtilde", vi)).l2NormSqr();
                // gravity
                E += -m * extForce.dot(x) * dt * dt;
              }

              // atomic_add(exec_cuda, &res[0], E);
              reduce_to(vi, numVerts, E, res[0]);
            });
        if (primHandle.category == ZenoParticles::surface)
          // elasticity
          pol(range(eles.size()),
              [eles = proxy<space>({}, eles), vtemp = proxy<space>({}, vtemp),
               res = proxy<space>(res), tag, model = model, dt = this->dt,
               vOffset = primHandle.vOffset,
               numEles = eles.size()] __device__(int ei) mutable {
                auto IB = eles.template pack<2, 2>("IB", ei);
                auto inds = eles.template pack<3>("inds", ei)
                                .template reinterpret_bits<int>() +
                            vOffset;

                T E;
                int BCorder[3];
                for (int i = 0; i != 3; ++i)
                  BCorder[i] = vtemp("BCorder", inds[i]);
                if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3)
                  E = 0;
                else {
                  auto vole = eles("vol", ei);
                  vec3 xs[3] = {vtemp.template pack<3>(tag, inds[0]),
                                vtemp.template pack<3>(tag, inds[1]),
                                vtemp.template pack<3>(tag, inds[2])};
                  auto x1x0 = xs[1] - xs[0];
                  auto x2x0 = xs[2] - xs[0];

                  zs::vec<T, 3, 2> Ds{x1x0[0], x2x0[0], x1x0[1],
                                      x2x0[1], x1x0[2], x2x0[2]};
                  auto F = Ds * IB;
                  auto f0 = col(F, 0);
                  auto f1 = col(F, 1);
                  auto f0Norm = zs::sqrt(f0.l2NormSqr());
                  auto f1Norm = zs::sqrt(f1.l2NormSqr());
                  auto Estretch = model.mu * vole *
                                  (zs::sqr(f0Norm - 1) + zs::sqr(f1Norm - 1));
                  auto Eshear = (model.mu * 0.3) * vole * zs::sqr(f0.dot(f1));
                  E = Estretch + Eshear;
                  E *= dt * dt;
                }

                // atomic_add(exec_cuda, &res[0], E);
                reduce_to(ei, numEles, E, res[0]);
              });
        else if (primHandle.category == ZenoParticles::tet)
          pol(zs::range(eles.size()),
              [vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, eles),
               res = proxy<space>(res), model, dt = this->dt,
               vOffset = primHandle.vOffset,
               numEles = eles.size()] __device__(int ei) mutable {
                auto IB = eles.template pack<3, 3>("IB", ei);
                auto inds = eles.template pack<4>("inds", ei)
                                .template reinterpret_bits<int>() +
                            vOffset;
                auto vole = eles("vol", ei);
                vec3 xs[4] = {
                    vtemp.pack<3>("xn", inds[0]), vtemp.pack<3>("xn", inds[1]),
                    vtemp.pack<3>("xn", inds[2]), vtemp.pack<3>("xn", inds[3])};

                int BCorder[4];
                for (int i = 0; i != 4; ++i)
                  BCorder[i] = vtemp("BCorder", inds[i]);

                T E;
                if (BCorder[0] == 3 && BCorder[1] == 3 && BCorder[2] == 3 &&
                    BCorder[3] == 3)
                  E = 0;
                else {
                  mat3 F{};
                  auto x1x0 = xs[1] - xs[0];
                  auto x2x0 = xs[2] - xs[0];
                  auto x3x0 = xs[3] - xs[0];
                  auto Ds = mat3{x1x0[0], x2x0[0], x3x0[0], x1x0[1], x2x0[1],
                                 x3x0[1], x1x0[2], x2x0[2], x3x0[2]};
                  F = Ds * IB;
                  E = model.psi(F) * dt * dt * vole;
                }
                reduce_to(ei, numEles, E, res[0]);
              });
      }
#if 0
      // collision object
      pol(range(coVerts.size()),
          [vtemp = proxy<space>({}, vtemp), res = proxy<space>(res), tag,
           projectDBC = projectDBC, dt = this->dt, coOffset = coOffset,
           n = coVerts.size()] __device__(int vi) mutable {
            vi += coOffset;
            T E;
            if (projectDBC || vtemp("BCfixed", vi) == 1)
              E = 0;
            else {
              // inertia
              auto m = zs::sqr(vtemp("ws", vi));
              auto x = vtemp.pack<3>(tag, vi);
              E = (T)0.5 * m * (x - vtemp.pack<3>("xtilde", vi)).l2NormSqr();
            };
            reduce_to(vi, n, E, res[0]);
          });
#endif
      // contacts
      {
#if s_enableContact
        {
          auto activeGap2 = dHat * dHat + 2 * xi * dHat;
          auto numPP = nPP.getVal();
          pol(range(numPP),
              [vtemp = proxy<space>({}, vtemp),
               tempPP = proxy<space>({}, tempPP), PP = proxy<space>(PP),
               res = proxy<space>(res), xi2 = xi * xi, dHat = dHat, activeGap2,
               kappa = kappa, numPP] __device__(int ppi) mutable {
                auto pp = PP[ppi];
                auto x0 = vtemp.pack<3>("xn", pp[0]);
                auto x1 = vtemp.pack<3>("xn", pp[1]);
                auto dist2 = dist2_pp(x0, x1);
                if (dist2 < xi2)
                  printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                reduce_to(ppi, numPP,
                          zs::barrier(dist2 - xi2, activeGap2, kappa), res[0]);
              });
          auto numPE = nPE.getVal();
          pol(range(numPE),
              [vtemp = proxy<space>({}, vtemp),
               tempPE = proxy<space>({}, tempPE), PE = proxy<space>(PE),
               res = proxy<space>(res), xi2 = xi * xi, dHat = dHat, activeGap2,
               kappa = kappa, numPE] __device__(int pei) mutable {
                auto pe = PE[pei];
                auto p = vtemp.pack<3>("xn", pe[0]);
                auto e0 = vtemp.pack<3>("xn", pe[1]);
                auto e1 = vtemp.pack<3>("xn", pe[2]);

                auto dist2 = dist2_pe(p, e0, e1);
                if (dist2 < xi2)
                  printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                reduce_to(pei, numPE,
                          zs::barrier(dist2 - xi2, activeGap2, kappa), res[0]);
              });
          auto numPT = nPT.getVal();
          pol(range(numPT),
              [vtemp = proxy<space>({}, vtemp),
               tempPT = proxy<space>({}, tempPT), PT = proxy<space>(PT),
               res = proxy<space>(res), xi2 = xi * xi, dHat = dHat, activeGap2,
               kappa = kappa, numPT] __device__(int pti) mutable {
                auto pt = PT[pti];
                auto p = vtemp.pack<3>("xn", pt[0]);
                auto t0 = vtemp.pack<3>("xn", pt[1]);
                auto t1 = vtemp.pack<3>("xn", pt[2]);
                auto t2 = vtemp.pack<3>("xn", pt[3]);

                auto dist2 = dist2_pt(p, t0, t1, t2);
                if (dist2 < xi2)
                  printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                reduce_to(pti, numPT,
                          zs::barrier(dist2 - xi2, activeGap2, kappa), res[0]);
              });
          auto numEE = nEE.getVal();
          pol(range(numEE),
              [vtemp = proxy<space>({}, vtemp),
               tempEE = proxy<space>({}, tempEE), EE = proxy<space>(EE),
               res = proxy<space>(res), xi2 = xi * xi, dHat = dHat, activeGap2,
               kappa = kappa, numEE] __device__(int eei) mutable {
                auto ee = EE[eei];
                auto ea0 = vtemp.pack<3>("xn", ee[0]);
                auto ea1 = vtemp.pack<3>("xn", ee[1]);
                auto eb0 = vtemp.pack<3>("xn", ee[2]);
                auto eb1 = vtemp.pack<3>("xn", ee[3]);

                auto dist2 = dist2_ee(ea0, ea1, eb0, eb1);
                if (dist2 < xi2)
                  printf("dist already smaller than xi!\n");
                // atomic_add(exec_cuda, &res[0],
                //           zs::barrier(dist2 - xi2, activeGap2, kappa));
                reduce_to(eei, numEE,
                          zs::barrier(dist2 - xi2, activeGap2, kappa), res[0]);
              });
        }
#endif
        // boundary
        pol(range(coOffset),
            [vtemp = proxy<space>({}, vtemp), res = proxy<space>(res),
             gn = s_groundNormal, dHat2 = dHat * dHat,
             kappa = kappa] ZS_LAMBDA(int vi) mutable {
              auto x = vtemp.pack<3>("xn", vi);
              auto dist = gn.dot(x);
              auto dist2 = dist * dist;
              if (dist2 < dHat2) {
                auto temp = -(dist2 - dHat2) * (dist2 - dHat2) *
                            zs::log(dist2 / dHat2) * kappa;
                atomic_add(exec_cuda, &res[0], temp);
              }
            });
      }
      // constraints
      if (includeAugLagEnergy) {
        computeConstraints(pol, tag);
        pol(range(numDofs), [vtemp = proxy<space>({}, vtemp),
                             res = proxy<space>(res),
                             kappa = kappa] __device__(int vi) mutable {
          // already updated during "xn" update
          auto cons = vtemp.template pack<3>("cons", vi);
          auto w = vtemp("ws", vi);
          auto lambda = vtemp.pack<3>("lambda", vi);
          atomic_add(
              exec_cuda, &res[0],
              (T)(-lambda.dot(cons) * w + 0.5 * kappa * w * cons.l2NormSqr()));
        });
      }
      return res.getVal();
    }
#endif
    void checkSPD(zs::CudaExecutionPolicy &pol,
                  const zs::SmallString dxTag) const {
      using namespace zs;
      constexpr execspace_e space = execspace_e::cuda;
      constexpr auto execTag = wrapv<space>{};
      auto checkHess = [] __device__(const auto &m,
                                     const zs::SmallString &msg = "") -> bool {
        auto checkDet = [&msg](auto &checkDet, const auto &m) -> bool {
          using MatT = RM_CVREF_T(m);
          using Ti = typename MatT::index_type;
          using T = typename MatT::value_type;
          constexpr int dim = MatT::template range_t<0>::value;
          if (auto det = determinant(m); det < -limits<T>::epsilon()) {
            printf("msg[%s]: subblock[%d] determinant is %f\n", msg.asChars(),
                   (int)dim, (float)det);
            return true;
          }
          if constexpr (dim > 1) {
            using SubMatT = typename MatT::template variant_vec<
                T, integer_seq<Ti, dim - 1, dim - 1>>;
            SubMatT subm;
            for (int i = 0; i != dim - 1; ++i)
              for (int j = 0; j != dim - 1; ++j)
                subm(i, j) = m(i, j);
            return checkDet(checkDet, subm);
          }
          return false;
        };
        return checkDet(checkDet, m);
      };
      // inertial
      pol(zs::range(coOffset), [checkHess,
                                tempPB = proxy<space>(
                                    {}, tempPB)] __device__(int i) mutable {
        auto Hi = tempPB.template pack<3, 3>("Hi", i);
        checkHess(Hi, "inertial");
        if (Hi(0, 0) < 0 || Hi(1, 1) < 0 || Hi(2, 2) < 0)
          printf(
              "%d-th Inertial Hessian [%f, %f, %f; %f, %f, %f; %f, %f, %f]\n",
              i, (float)Hi(0, 0), (float)Hi(0, 1), (float)Hi(0, 2),
              (float)Hi(1, 0), (float)Hi(1, 1), (float)Hi(1, 2),
              (float)Hi(2, 0), (float)Hi(2, 1), (float)Hi(2, 2));
      });

      for (auto &primHandle : prims) {
        auto &verts = primHandle.getVerts();
        auto &eles = primHandle.getEles();
        // elasticity
        if (primHandle.category == ZenoParticles::surface) {
          pol(range(eles.size()),
              [checkHess, etemp = proxy<space>({}, primHandle.etemp),
               vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, eles),
               vOffset = primHandle.vOffset] ZS_LAMBDA(int ei) mutable {
                constexpr int dim = 3;
                auto inds = eles.template pack<3>("inds", ei)
                                .template reinterpret_bits<int>() +
                            vOffset;
                auto He = etemp.template pack<dim * 3, dim * 3>("He", ei);
                checkHess(He, "surf elasticity");
                for (int d = 0; d != 3; ++d) {
                  mat3 Hi{};
                  for (int e = 0; e != 9; ++e)
                    Hi(e / 3, e % 3) = He(d * 3 + e / 3, d * 3 + e % 3);

                  if (Hi(0, 0) < 0 || Hi(1, 1) < 0 || Hi(2, 2) < 0)
                    printf("%d-th Elastic Hessian9 %d-th subdiagblock:\n\t[%f, "
                           "%f, %f; %f, %f, %f; %f, "
                           "%f, %f]\n",
                           ei, d, (float)Hi(0, 0), (float)Hi(0, 1),
                           (float)Hi(0, 2), (float)Hi(1, 0), (float)Hi(1, 1),
                           (float)Hi(1, 2), (float)Hi(2, 0), (float)Hi(2, 1),
                           (float)Hi(2, 2));
                }
              });
        } else if (primHandle.category == ZenoParticles::tet)
          pol(range(eles.size()),
              [checkHess, etemp = proxy<space>({}, primHandle.etemp),
               vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, eles),
               vOffset = primHandle.vOffset] ZS_LAMBDA(int ei) mutable {
                constexpr int dim = 3;
                auto inds = eles.template pack<4>("inds", ei)
                                .template reinterpret_bits<int>() +
                            vOffset;
                auto He = etemp.template pack<dim * 4, dim * 4>("He", ei);
                checkHess(He, "tet elasticity");
                for (int d = 0; d != 4; ++d) {
                  mat3 Hi{};
                  for (int e = 0; e != 9; ++e)
                    Hi(e / 3, e % 3) = He(d * 3 + e / 3, d * 3 + e % 3);

                  if (Hi(0, 0) < 0 || Hi(1, 1) < 0 || Hi(2, 2) < 0)
                    printf(
                        "%d-th Elastic Hessian12 %d-th subdiagblock:\n\t[%f, "
                        "%f, %f; %f, %f, %f; %f, "
                        "%f, %f]\n",
                        ei, d, (float)Hi(0, 0), (float)Hi(0, 1),
                        (float)Hi(0, 2), (float)Hi(1, 0), (float)Hi(1, 1),
                        (float)Hi(1, 2), (float)Hi(2, 0), (float)Hi(2, 1),
                        (float)Hi(2, 2));
                }
              });
      }
      // contacts
      {
#if s_enableContact
        {
          auto numPP = nPP.getVal();
          pol(range(numPP), [checkHess, tempPP = proxy<space>({}, tempPP),
                             vtemp = proxy<space>({}, vtemp),
                             PP = proxy<space>(PP)] ZS_LAMBDA(int ppi) mutable {
            constexpr int dim = 3;
            auto pp = PP[ppi];
            auto ppHess = tempPP.template pack<6, 6>("H", ppi);
            checkHess(ppHess, "pp hess");
            for (int d = 0; d != 2; ++d) {
              mat3 Hi{};
              for (int e = 0; e != 9; ++e)
                Hi(e / 3, e % 3) = ppHess(d * 3 + e / 3, d * 3 + e % 3);

              if (Hi(0, 0) < 0 || Hi(1, 1) < 0 || Hi(2, 2) < 0)
                printf("%d-th Contact Hessian6 %d-th subdiagblock:\n\t[%f, "
                       "%f, %f; %f, %f, %f; %f, "
                       "%f, %f]\n",
                       ppi, d, (float)Hi(0, 0), (float)Hi(0, 1),
                       (float)Hi(0, 2), (float)Hi(1, 0), (float)Hi(1, 1),
                       (float)Hi(1, 2), (float)Hi(2, 0), (float)Hi(2, 1),
                       (float)Hi(2, 2));
            }
          });
          auto numPE = nPE.getVal();
          pol(range(numPE), [checkHess, tempPE = proxy<space>({}, tempPE),
                             vtemp = proxy<space>({}, vtemp),
                             PE = proxy<space>(PE)] ZS_LAMBDA(int pei) mutable {
            constexpr int dim = 3;
            auto pe = PE[pei];
            auto peHess = tempPE.template pack<9, 9>("H", pei);
            checkHess(peHess, "pe hess");
            for (int d = 0; d != 3; ++d) {
              mat3 Hi{};
              for (int e = 0; e != 9; ++e)
                Hi(e / 3, e % 3) = peHess(d * 3 + e / 3, d * 3 + e % 3);

              if (Hi(0, 0) < 0 || Hi(1, 1) < 0 || Hi(2, 2) < 0)
                printf("%d-th Contact Hessian9 %d-th subdiagblock:\n\t[%f, "
                       "%f, %f; %f, %f, %f; %f, "
                       "%f, %f]\n",
                       pei, d, (float)Hi(0, 0), (float)Hi(0, 1),
                       (float)Hi(0, 2), (float)Hi(1, 0), (float)Hi(1, 1),
                       (float)Hi(1, 2), (float)Hi(2, 0), (float)Hi(2, 1),
                       (float)Hi(2, 2));
            }
          });
          auto numPT = nPT.getVal();
          pol(range(numPT), [checkHess, tempPT = proxy<space>({}, tempPT),
                             vtemp = proxy<space>({}, vtemp),
                             PT = proxy<space>(PT)] ZS_LAMBDA(int pti) mutable {
            constexpr int dim = 3;
            auto pt = PT[pti];
            auto ptHess = tempPT.template pack<12, 12>("H", pti);
            checkHess(ptHess, "pt hess");
            for (int d = 0; d != 4; ++d) {
              mat3 Hi{};
              for (int e = 0; e != 9; ++e)
                Hi(e / 3, e % 3) = ptHess(d * 3 + e / 3, d * 3 + e % 3);

              if (Hi(0, 0) < 0 || Hi(1, 1) < 0 || Hi(2, 2) < 0)
                printf("%d-th Contact Hessian12 %d-th subdiagblock:\n\t[%f, "
                       "%f, %f; %f, %f, %f; %f, "
                       "%f, %f]\n",
                       pti, d, (float)Hi(0, 0), (float)Hi(0, 1),
                       (float)Hi(0, 2), (float)Hi(1, 0), (float)Hi(1, 1),
                       (float)Hi(1, 2), (float)Hi(2, 0), (float)Hi(2, 1),
                       (float)Hi(2, 2));
            }
          });
          auto numEE = nEE.getVal();
          pol(range(numEE), [checkHess, tempEE = proxy<space>({}, tempEE),
                             vtemp = proxy<space>({}, vtemp),
                             EE = proxy<space>(EE)] ZS_LAMBDA(int eei) mutable {
            constexpr int dim = 3;
            auto ee = EE[eei];
            auto eeHess = tempEE.template pack<12, 12>("H", eei);
            if (checkHess(eeHess, "ee hess")) {
              auto ea0 = vtemp.template pack<3>("xn", ee[0]);
              auto ea1 = vtemp.template pack<3>("xn", ee[1]);
              auto eb0 = vtemp.template pack<3>("xn", ee[2]);
              auto eb1 = vtemp.template pack<3>("xn", ee[3]);
              auto ea0Rest = vtemp.template pack<3>("x0", ee[0]);
              auto ea1Rest = vtemp.template pack<3>("x0", ee[1]);
              auto eb0Rest = vtemp.template pack<3>("x0", ee[2]);
              auto eb1Rest = vtemp.template pack<3>("x0", ee[3]);
              auto cro = (ea1 - ea0).cross(eb1 - eb0);
              T c = cn2_ee(ea0, ea1, eb0, eb1);
              T epsX =
                  mollifier_threshold_ee(ea0Rest, ea1Rest, eb0Rest, eb1Rest);
              bool mollify = c < epsX;
              printf("e0 (%f, %f, %f)-(%f, %f, %f), e1 (%f, %f, %f)-(%f, %f, "
                     "%f), c (%f) < epsX (%f) ?\n",
                     (float)ea0[0], (float)ea0[1], (float)ea0[2], (float)ea1[0],
                     (float)ea1[1], (float)ea1[2], (float)eb0[0], (float)eb0[1],
                     (float)eb0[2], (float)eb1[0], (float)eb1[1], (float)eb1[2],
                     (float)c, (float)epsX);
            }
            for (int d = 0; d != 4; ++d) {
              mat3 Hi{};
              for (int e = 0; e != 9; ++e)
                Hi(e / 3, e % 3) = eeHess(d * 3 + e / 3, d * 3 + e % 3);

              if (Hi(0, 0) < 0 || Hi(1, 1) < 0 || Hi(2, 2) < 0)
                printf("%d-th Contact Hessian12 %d-th subdiagblock:\n\t[%f, "
                       "%f, %f; %f, %f, %f; %f, "
                       "%f, %f]\n",
                       eei, d, (float)Hi(0, 0), (float)Hi(0, 1),
                       (float)Hi(0, 2), (float)Hi(1, 0), (float)Hi(1, 1),
                       (float)Hi(1, 2), (float)Hi(2, 0), (float)Hi(2, 1),
                       (float)Hi(2, 2));
            }
          });
        }
#endif
      } // end contacts
      puts("done checking SPD");
    }
    void project(zs::CudaExecutionPolicy &pol, const zs::SmallString tag) {
      using namespace zs;
      constexpr execspace_e space = execspace_e::cuda;
      // projection
      pol(zs::range(numDofs),
          [vtemp = proxy<space>({}, vtemp), projectDBC = projectDBC,
           tag] ZS_LAMBDA(int vi) mutable {
            int BCfixed = vtemp("BCfixed", vi);
            if (projectDBC || (!projectDBC && BCfixed)) {
              int BCorder = vtemp("BCorder", vi);
              for (int d = 0; d != BCorder; ++d)
                vtemp(tag, d, vi) = 0;
            }
          });
    }
    void precondition(zs::CudaExecutionPolicy &pol,
                      const zs::SmallString srcTag,
                      const zs::SmallString dstTag) {
      using namespace zs;
      constexpr execspace_e space = execspace_e::cuda;
      // precondition
      pol(zs::range(numDofs), [vtemp = proxy<space>({}, vtemp), srcTag,
                               dstTag] ZS_LAMBDA(int vi) mutable {
        vtemp.template tuple<3>(dstTag, vi) =
            vtemp.template pack<3, 3>("P", vi) *
            vtemp.template pack<3>(srcTag, vi);
      });
    }
    void multiply(zs::CudaExecutionPolicy &pol, const zs::SmallString dxTag,
                  const zs::SmallString bTag) {
      using namespace zs;
      constexpr execspace_e space = execspace_e::cuda;
      constexpr auto execTag = wrapv<space>{};
      // hessian rotation: trans^T hess * trans
      // left trans^T: multiplied on rows
      // right trans: multiplied on cols
      // dx -> b
      pol(range(numDofs), [execTag, vtemp = proxy<space>({}, vtemp),
                           bTag] ZS_LAMBDA(int vi) mutable {
        vtemp.template tuple<3>(bTag, vi) = vec3::zeros();
      });
      // inertial
      pol(zs::range(coOffset), [execTag, tempPB = proxy<space>({}, tempPB),
                                vtemp = proxy<space>({}, vtemp), dxTag,
                                bTag] __device__(int i) mutable {
        auto Hi = tempPB.template pack<3, 3>("Hi", i);
        auto dx = vtemp.template pack<3>(dxTag, i);
        dx = Hi * dx;
        for (int d = 0; d != 3; ++d)
          atomic_add(execTag, &vtemp(bTag, d, i), dx(d));
      });

      for (auto &primHandle : prims) {
        auto &verts = primHandle.getVerts();
        auto &eles = primHandle.getEles();
        // elasticity
        if (primHandle.category == ZenoParticles::surface) {
#if 1
          pol(range(eles.size()),
              [execTag, etemp = proxy<space>({}, primHandle.etemp),
               vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, eles),
               dxTag, bTag,
               vOffset = primHandle.vOffset] ZS_LAMBDA(int ei) mutable {
                constexpr int dim = 3;
                auto inds = eles.template pack<3>("inds", ei)
                                .template reinterpret_bits<int>() +
                            vOffset;
                zs::vec<T, 3 * dim> temp{};
                for (int vi = 0; vi != 3; ++vi)
                  for (int d = 0; d != dim; ++d) {
                    temp[vi * dim + d] = vtemp(dxTag, d, inds[vi]);
                  }
                auto He = etemp.template pack<dim * 3, dim * 3>("He", ei);

                temp = He * temp;

                for (int vi = 0; vi != 3; ++vi)
                  for (int d = 0; d != dim; ++d) {
                    atomic_add(execTag, &vtemp(bTag, d, inds[vi]),
                               temp[vi * dim + d]);
                  }
              });
#else
          pol(range(eles.size() * 81),
              [execTag, etemp = proxy<space>({}, primHandle.etemp),
               vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, eles),
               dxTag, bTag, vOffset = primHandle.vOffset,
               n = eles.size() * 81] ZS_LAMBDA(int idx) mutable {
                constexpr int dim = 3;
                __shared__ int offset;
                // directly use PCG_Solve_AX9_b2 from kemeng huang
                int ei = idx / 81;
                int entryId = idx % 81;
                int MRid = entryId / 9;
                int MCid = entryId % 9;
                int vId = MCid / dim;
                int axisId = MCid % dim;
                int GRtid = idx % 9;

                auto inds = eles.template pack<3>("inds", ei)
                                .template reinterpret_bits<int>() +
                            vOffset;
                T rdata =
                    etemp("He", entryId, ei) * vtemp(dxTag, axisId, inds[vId]);

                if (threadIdx.x == 0)
                  offset = 9 - GRtid;
                __syncthreads();

                int BRid = (threadIdx.x - offset + 9) / 9;
                int landidx = (threadIdx.x - offset) % 9;
                if (BRid == 0) {
                  landidx = threadIdx.x;
                }

                auto [mask, numValid] = warp_mask(idx, n);
                int laneId = threadIdx.x & 0x1f;
                bool bBoundary = (landidx == 0) || (laneId == 0);

                unsigned int mark =
                    __ballot_sync(mask, bBoundary); // a bit-mask
                mark = __brev(mark);
                unsigned int interval =
                    zs::math::min(__clz(mark << (laneId + 1)), 31 - laneId);

                for (int iter = 1; iter < 9; iter <<= 1) {
                  T tmp = __shfl_down_sync(mask, rdata, iter);
                  if (interval >= iter && laneId + iter < numValid)
                    rdata += tmp;
                }

                if (bBoundary)
                  atomic_add(exec_cuda, &vtemp(bTag, MRid % 3, inds[MRid / 3]),
                             rdata);
              });
#endif
        } else if (primHandle.category == ZenoParticles::tet)
#if 1
          pol(range(eles.size()),
              [execTag, etemp = proxy<space>({}, primHandle.etemp),
               vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, eles),
               dxTag, bTag,
               vOffset = primHandle.vOffset] ZS_LAMBDA(int ei) mutable {
                constexpr int dim = 3;
                auto inds = eles.template pack<4>("inds", ei)
                                .template reinterpret_bits<int>() +
                            vOffset;
                zs::vec<T, 4 * dim> temp{};
                for (int vi = 0; vi != 4; ++vi)
                  for (int d = 0; d != dim; ++d) {
                    temp[vi * dim + d] = vtemp(dxTag, d, inds[vi]);
                  }
                auto He = etemp.template pack<dim * 4, dim * 4>("He", ei);

                temp = He * temp;

                for (int vi = 0; vi != 4; ++vi)
                  for (int d = 0; d != dim; ++d) {
                    atomic_add(execTag, &vtemp(bTag, d, inds[vi]),
                               temp[vi * dim + d]);
                  }
              });
#else
          pol(range(eles.size() * 144),
              [execTag, etemp = proxy<space>({}, primHandle.etemp),
               vtemp = proxy<space>({}, vtemp), eles = proxy<space>({}, eles),
               dxTag, bTag, vOffset = primHandle.vOffset,
               n = eles.size() * 144] ZS_LAMBDA(int idx) mutable {
                constexpr int dim = 3;
                __shared__ int offset;
                // directly use PCG_Solve_AX9_b2 from kemeng huang
                int Hid = idx / 144;
                int entryId = idx % 144;
                int MRid = entryId / 12;
                int MCid = entryId % 12;
                int vId = MCid / dim;
                int axisId = MCid % dim;
                int GRtid = idx % 12;

                auto inds = eles.template pack<4>("inds", Hid)
                                .template reinterpret_bits<int>() +
                            vOffset;
                T rdata =
                    etemp("He", entryId, Hid) * vtemp(dxTag, axisId, inds[vId]);

                if (threadIdx.x == 0)
                  offset = 12 - GRtid;
                __syncthreads();

                int BRid = (threadIdx.x - offset + 12) / 12;
                int landidx = (threadIdx.x - offset) % 12;
                if (BRid == 0) {
                  landidx = threadIdx.x;
                }

                auto [mask, numValid] = warp_mask(idx, n);
                int laneId = threadIdx.x & 0x1f;
                bool bBoundary = (landidx == 0) || (laneId == 0);

                unsigned int mark =
                    __ballot_sync(mask, bBoundary); // a bit-mask
                mark = __brev(mark);
                unsigned int interval =
                    zs::math::min(__clz(mark << (laneId + 1)), 31 - laneId);

                for (int iter = 1; iter < 12; iter <<= 1) {
                  T tmp = __shfl_down_sync(mask, rdata, iter);
                  if (interval >= iter && laneId + iter < numValid)
                    rdata += tmp;
                }

                if (bBoundary)
                  atomic_add(exec_cuda, &vtemp(bTag, MRid % 3, inds[MRid / 3]),
                             rdata);
              });
#endif
      }
#if 0
      // collision object
      pol(range(coVerts.size()),
          [execTag, verts = proxy<space>({}, coVerts),
           vtemp = proxy<space>({}, vtemp), projectDBC = projectDBC, dxTag,
           bTag, coOffset = coOffset] ZS_LAMBDA(int vi) mutable {
            vi += coOffset;
            if (projectDBC || vtemp("BCfixed", vi) == 1)
              return;
            // inertia
            auto m = zs::sqr(vtemp("ws", vi));
            auto dx = vtemp.template pack<3>(dxTag, vi);
            dx = m * dx;
            for (int d = 0; d != 3; ++d)
              atomic_add(execTag, &vtemp(bTag, d, vi), dx(d));
          });
#endif
      // contacts
      {
#if s_enableContact
        {
          auto numPP = nPP.getVal();
#if 1
          pol(range(numPP), [execTag, tempPP = proxy<space>({}, tempPP),
                             vtemp = proxy<space>({}, vtemp), dxTag, bTag,
                             PP = proxy<space>(PP)] ZS_LAMBDA(int ppi) mutable {
            constexpr int dim = 3;
            auto pp = PP[ppi];
            zs::vec<T, dim * 2> temp{};
            for (int vi = 0; vi != 2; ++vi)
              for (int d = 0; d != dim; ++d) {
                temp[vi * dim + d] = vtemp(dxTag, d, pp[vi]);
              }
            auto ppHess = tempPP.template pack<6, 6>("H", ppi);

            temp = ppHess * temp;

            for (int vi = 0; vi != 2; ++vi)
              for (int d = 0; d != dim; ++d) {
                atomic_add(execTag, &vtemp(bTag, d, pp[vi]),
                           temp[vi * dim + d]);
              }
          });
#else
          pol(range(numPP * 36), [execTag, tempPP = proxy<space>({}, tempPP),
                                  vtemp = proxy<space>({}, vtemp), dxTag, bTag,
                                  PP = proxy<space>(PP),
                                  n = numPP * 36] ZS_LAMBDA(int idx) mutable {
            constexpr int dim = 3;
            __shared__ int offset;
            // directly use PCG_Solve_AX9_b2 from kemeng huang
            int Hid = idx / 36;
            int entryId = idx % 36;
            int MRid = entryId / 6;
            int MCid = entryId % 6;
            int vId = MCid / dim;
            int axisId = MCid % dim;
            int GRtid = idx % 6;

            auto inds = PP[Hid];
            T rdata =
                tempPP("H", entryId, Hid) * vtemp(dxTag, axisId, inds[vId]);

            if (threadIdx.x == 0)
              offset = 6 - GRtid;
            __syncthreads();

            int BRid = (threadIdx.x - offset + 6) / 6;
            int landidx = (threadIdx.x - offset) % 6;
            if (BRid == 0) {
              landidx = threadIdx.x;
            }

            auto [mask, numValid] = warp_mask(idx, n);
            int laneId = threadIdx.x & 0x1f;
            bool bBoundary = (landidx == 0) || (laneId == 0);

            unsigned int mark = __ballot_sync(mask, bBoundary); // a bit-mask
            mark = __brev(mark);
            unsigned int interval =
                zs::math::min(__clz(mark << (laneId + 1)), 31 - laneId);

            for (int iter = 1; iter < 6; iter <<= 1) {
              T tmp = __shfl_down_sync(mask, rdata, iter);
              if (interval >= iter && laneId + iter < numValid)
                rdata += tmp;
            }

            if (bBoundary)
              atomic_add(exec_cuda, &vtemp(bTag, MRid % 3, inds[MRid / 3]),
                         rdata);
          });
#endif
          auto numPE = nPE.getVal();
#if 1
          pol(range(numPE), [execTag, tempPE = proxy<space>({}, tempPE),
                             vtemp = proxy<space>({}, vtemp), dxTag, bTag,
                             PE = proxy<space>(PE)] ZS_LAMBDA(int pei) mutable {
            constexpr int dim = 3;
            auto pe = PE[pei];
            zs::vec<T, dim * 3> temp{};
            for (int vi = 0; vi != 3; ++vi)
              for (int d = 0; d != dim; ++d) {
                temp[vi * dim + d] = vtemp(dxTag, d, pe[vi]);
              }
            auto peHess = tempPE.template pack<9, 9>("H", pei);

            temp = peHess * temp;

            for (int vi = 0; vi != 3; ++vi)
              for (int d = 0; d != dim; ++d) {
                atomic_add(execTag, &vtemp(bTag, d, pe[vi]),
                           temp[vi * dim + d]);
              }
          });
#else
          pol(range(numPE * 81), [execTag, tempPE = proxy<space>({}, tempPE),
                                  vtemp = proxy<space>({}, vtemp), dxTag, bTag,
                                  PE = proxy<space>(PE),
                                  n = numPE * 81] ZS_LAMBDA(int idx) mutable {
            constexpr int dim = 3;
            __shared__ int offset;
            // directly use PCG_Solve_AX9_b2 from kemeng huang
            int Hid = idx / 81;
            int entryId = idx % 81;
            int MRid = entryId / 9;
            int MCid = entryId % 9;
            int vId = MCid / dim;
            int axisId = MCid % dim;
            int GRtid = idx % 9;

            auto inds = PE[Hid];
            T rdata =
                tempPE("H", entryId, Hid) * vtemp(dxTag, axisId, inds[vId]);

            if (threadIdx.x == 0)
              offset = 9 - GRtid;
            __syncthreads();

            int BRid = (threadIdx.x - offset + 9) / 9;
            int landidx = (threadIdx.x - offset) % 9;
            if (BRid == 0) {
              landidx = threadIdx.x;
            }

            auto [mask, numValid] = warp_mask(idx, n);
            int laneId = threadIdx.x & 0x1f;
            bool bBoundary = (landidx == 0) || (laneId == 0);

            unsigned int mark = __ballot_sync(mask, bBoundary); // a bit-mask
            mark = __brev(mark);
            unsigned int interval =
                zs::math::min(__clz(mark << (laneId + 1)), 31 - laneId);

            for (int iter = 1; iter < 9; iter <<= 1) {
              T tmp = __shfl_down_sync(mask, rdata, iter);
              if (interval >= iter && laneId + iter < numValid)
                rdata += tmp;
            }

            if (bBoundary)
              atomic_add(exec_cuda, &vtemp(bTag, MRid % 3, inds[MRid / 3]),
                         rdata);
          });
#endif
          auto numPT = nPT.getVal();
#if 1
          pol(range(numPT), [execTag, tempPT = proxy<space>({}, tempPT),
                             vtemp = proxy<space>({}, vtemp), dxTag, bTag,
                             PT = proxy<space>(PT)] ZS_LAMBDA(int pti) mutable {
            constexpr int dim = 3;
            auto pt = PT[pti];
            zs::vec<T, dim * 4> temp{};
            for (int vi = 0; vi != 4; ++vi)
              for (int d = 0; d != dim; ++d) {
                temp[vi * dim + d] = vtemp(dxTag, d, pt[vi]);
              }
            auto ptHess = tempPT.template pack<12, 12>("H", pti);

            temp = ptHess * temp;

            for (int vi = 0; vi != 4; ++vi)
              for (int d = 0; d != dim; ++d) {
                atomic_add(execTag, &vtemp(bTag, d, pt[vi]),
                           temp[vi * dim + d]);
              }
          });
#else
          pol(range(numPT * 144), [execTag, tempPT = proxy<space>({}, tempPT),
                                   vtemp = proxy<space>({}, vtemp), dxTag, bTag,
                                   PT = proxy<space>(PT),
                                   n = numPT * 144] ZS_LAMBDA(int idx) mutable {
            constexpr int dim = 3;
            __shared__ int offset;
            // directly use PCG_Solve_AX9_b2 from kemeng huang
            int Hid = idx / 144;
            int entryId = idx % 144;
            int MRid = entryId / 12;
            int MCid = entryId % 12;
            int vId = MCid / dim;
            int axisId = MCid % dim;
            int GRtid = idx % 12;

            auto inds = PT[Hid];
            T rdata =
                tempPT("H", entryId, Hid) * vtemp(dxTag, axisId, inds[vId]);

            if (threadIdx.x == 0)
              offset = 12 - GRtid;
            __syncthreads();

            int BRid = (threadIdx.x - offset + 12) / 12;
            int landidx = (threadIdx.x - offset) % 12;
            if (BRid == 0) {
              landidx = threadIdx.x;
            }

            auto [mask, numValid] = warp_mask(idx, n);
            int laneId = threadIdx.x & 0x1f;
            bool bBoundary = (landidx == 0) || (laneId == 0);

            unsigned int mark = __ballot_sync(mask, bBoundary); // a bit-mask
            mark = __brev(mark);
            unsigned int interval =
                zs::math::min(__clz(mark << (laneId + 1)), 31 - laneId);

            for (int iter = 1; iter < 12; iter <<= 1) {
              T tmp = __shfl_down_sync(mask, rdata, iter);
              if (interval >= iter && laneId + iter < numValid)
                rdata += tmp;
            }

            if (bBoundary)
              atomic_add(exec_cuda, &vtemp(bTag, MRid % 3, inds[MRid / 3]),
                         rdata);
          });
#endif
          auto numEE = nEE.getVal();
#if 1
          pol(range(numEE), [execTag, tempEE = proxy<space>({}, tempEE),
                             vtemp = proxy<space>({}, vtemp), dxTag, bTag,
                             EE = proxy<space>(EE)] ZS_LAMBDA(int eei) mutable {
            constexpr int dim = 3;
            auto ee = EE[eei];
            zs::vec<T, dim * 4> temp{};
            for (int vi = 0; vi != 4; ++vi)
              for (int d = 0; d != dim; ++d) {
                temp[vi * dim + d] = vtemp(dxTag, d, ee[vi]);
              }
            auto eeHess = tempEE.template pack<12, 12>("H", eei);

            temp = eeHess * temp;

            for (int vi = 0; vi != 4; ++vi)
              for (int d = 0; d != dim; ++d) {
                atomic_add(execTag, &vtemp(bTag, d, ee[vi]),
                           temp[vi * dim + d]);
              }
          });
#else
          pol(range(numEE * 144), [execTag, tempEE = proxy<space>({}, tempEE),
                                   vtemp = proxy<space>({}, vtemp), dxTag, bTag,
                                   EE = proxy<space>(EE),
                                   n = numEE * 144] ZS_LAMBDA(int idx) mutable {
            constexpr int dim = 3;
            __shared__ int offset;
            // directly use PCG_Solve_AX9_b2 from kemeng huang
            int Hid = idx / 144;
            int entryId = idx % 144;
            int MRid = entryId / 12;
            int MCid = entryId % 12;
            int vId = MCid / dim;
            int axisId = MCid % dim;
            int GRtid = idx % 12;

            auto inds = EE[Hid];
            T rdata =
                tempEE("H", entryId, Hid) * vtemp(dxTag, axisId, inds[vId]);

            if (threadIdx.x == 0)
              offset = 12 - GRtid;
            __syncthreads();

            int BRid = (threadIdx.x - offset + 12) / 12;
            int landidx = (threadIdx.x - offset) % 12;
            if (BRid == 0) {
              landidx = threadIdx.x;
            }

            auto [mask, numValid] = warp_mask(idx, n);
            int laneId = threadIdx.x & 0x1f;
            bool bBoundary = (landidx == 0) || (laneId == 0);

            unsigned int mark = __ballot_sync(mask, bBoundary); // a bit-mask
            mark = __brev(mark);
            unsigned int interval =
                zs::math::min(__clz(mark << (laneId + 1)), 31 - laneId);

            for (int iter = 1; iter < 12; iter <<= 1) {
              T tmp = __shfl_down_sync(mask, rdata, iter);
              if (interval >= iter && laneId + iter < numValid)
                rdata += tmp;
            }

            if (bBoundary)
              atomic_add(exec_cuda, &vtemp(bTag, MRid % 3, inds[MRid / 3]),
                         rdata);
          });
#endif
        }
#endif
        // boundary
        pol(range(coOffset), [execTag, vtemp = proxy<space>({}, vtemp),
                              tempPB = proxy<space>({}, tempPB), dxTag,
                              bTag] ZS_LAMBDA(int vi) mutable {
          auto dx = vtemp.template pack<3>(dxTag, vi);
          auto pbHess = tempPB.template pack<3, 3>("H", vi);
          dx = pbHess * dx;
          for (int d = 0; d != 3; ++d)
            atomic_add(execTag, &vtemp(bTag, d, vi), dx(d));
        });
      } // end contacts

      // constraint hessian
      if (!BCsatisfied) {
        pol(range(numDofs), [execTag, vtemp = proxy<space>({}, vtemp), dxTag,
                             bTag, kappa = kappa] ZS_LAMBDA(int vi) mutable {
          auto cons = vtemp.template pack<3>("cons", vi);
          auto dx = vtemp.template pack<3>(dxTag, vi);
          auto w = vtemp("ws", vi);
          for (int d = 0; d != 3; ++d)
            if (cons[d] != 0)
              atomic_add(execTag, &vtemp(bTag, d, vi), kappa * w * dx(d));
        });
      }
    }
    void cgsolve(zs::CudaExecutionPolicy &cudaPol, bool &useGD) {
      // input "grad", multiply, constraints
      // output "dir"
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      if (useGD) {
        // project(cudaPol, "grad");
        precondition(cudaPol, "grad", "dir");
      } else {
        // solve for A dir = grad;
        cudaPol(zs::range(numDofs),
                [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
                  vtemp.tuple<3>("dir", i) = vec3::zeros();
                });
        // temp = A * dir
        multiply(cudaPol, "dir", "temp");
        // r = grad - temp
        cudaPol(zs::range(numDofs),
                [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
                  vtemp.tuple<3>("r", i) =
                      vtemp.pack<3>("grad", i) - vtemp.pack<3>("temp", i);
                });
        // project(cudaPol, "r");
        precondition(cudaPol, "r", "q");
        cudaPol(zs::range(numDofs),
                [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
                  vtemp.tuple<3>("p", i) = vtemp.pack<3>("q", i);
                });
        T zTrk = dot(cudaPol, vtemp, "r", "q");
        auto residualPreconditionedNorm = std::sqrt(zTrk);
        auto localTol = cgRel * residualPreconditionedNorm;
        int iter = 0;

        //
        auto [npp, npe, npt, nee, ncspt, ncsee] = getCnts();

        for (; iter != CGCap; ++iter) {
          if (iter % 25 == 0)
            fmt::print("cg iter: {}, norm: {} (zTrk: {}) npp: {}, npe: {}, "
                       "npt: {}, nee: {}, ncspt: {}, ncsee: {}\n",
                       iter, residualPreconditionedNorm, zTrk, npp, npe, npt,
                       nee, ncspt, ncsee);

          if (residualPreconditionedNorm <= localTol)
            break;
          multiply(cudaPol, "p", "temp");
          // project(cudaPol, "temp");

          T alpha = zTrk / dot(cudaPol, vtemp, "temp", "p");
          cudaPol(range(numDofs), [vtemp = proxy<space>({}, vtemp),
                                   alpha] ZS_LAMBDA(int vi) mutable {
            vtemp.tuple<3>("dir", vi) =
                vtemp.pack<3>("dir", vi) + alpha * vtemp.pack<3>("p", vi);
            vtemp.tuple<3>("r", vi) =
                vtemp.pack<3>("r", vi) - alpha * vtemp.pack<3>("temp", vi);
          });

          precondition(cudaPol, "r", "q");
          auto zTrkLast = zTrk;
          zTrk = dot(cudaPol, vtemp, "q", "r");
          auto beta = zTrk / zTrkLast;
          cudaPol(range(numDofs), [vtemp = proxy<space>({}, vtemp),
                                   beta] ZS_LAMBDA(int vi) mutable {
            vtemp.tuple<3>("p", vi) =
                vtemp.pack<3>("q", vi) + beta * vtemp.pack<3>("p", vi);
          });

          if (zTrk < 0) {
            fmt::print(fg(fmt::color::pale_violet_red),
                       "what the heck? zTrk: {} at iteration {}. switching to "
                       "gradient descent ftm.\n",
                       zTrk, iter);
            useGD = true;
            checkSPD(cudaPol, "xn");
            getchar();
            break;
          }
          residualPreconditionedNorm = std::sqrt(zTrk);
        } // end cg step
        if (useGD == true)
          return;
      }
    }
    void lineSearch(zs::CudaExecutionPolicy &cudaPol, T &alpha,
                    bool CCDfiltered) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      // initial energy
      T E0{};
      match([&](auto &elasticModel) {
        E0 = energy(cudaPol, elasticModel, "xn0", !BCsatisfied);
      })(models.getElasticModel());

      T E{E0};
      T c1m = 0;
      int lsIter = 0;
      c1m = armijoParam * dot(cudaPol, vtemp, "dir", "grad");
      fmt::print(fg(fmt::color::white), "c1m : {}\n", c1m);
#if 1
      do {
        cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp),
                                          alpha] __device__(int i) mutable {
          vtemp.tuple<3>("xn", i) =
              vtemp.pack<3>("xn0", i) + alpha * vtemp.pack<3>("dir", i);
        });

        if constexpr (s_enableContact)
          findCollisionConstraints(cudaPol, dHat, xi, s_enableAdaptiveSetting);
        match([&](auto &elasticModel) {
          E = energy(cudaPol, elasticModel, "xn", !BCsatisfied);
        })(models.getElasticModel());

        fmt::print("E: {} at alpha {}. E0 {}\n", E, alpha, E0);
#if 0
        if (E < E0) break;
#else
        if (E <= E0 + alpha * c1m)
          break;
#endif

        alpha /= 2;
        if (++lsIter > 30) {
          auto cr = constraintResidual(cudaPol);
          fmt::print(
              "too small stepsize at iteration [{}]! alpha: {}, cons res: {}\n",
              lsIter, alpha, cr);
          if (!useGD && !CCDfiltered)
            getchar();
        }
      } while (true);
#endif
    }
    void initialize(zs::CudaExecutionPolicy &pol) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      stInds = tiles_t{vtemp.get_allocator(), {{"inds", 3}}, sfOffset};
      seInds = tiles_t{vtemp.get_allocator(), {{"inds", 2}}, seOffset};
      svInds = tiles_t{vtemp.get_allocator(), {{"inds", 1}}, svOffset};
      for (auto &primHandle : prims) {
        auto &verts = primHandle.getVerts();
        // initialize BC info
        // predict pos, initialize augmented lagrangian, constrain weights
        pol(Collapse(verts.size()),
            [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts),
             voffset = primHandle.vOffset, dt = dt,
             extForce = extForce] __device__(int i) mutable {
              int BCorder = verts("BCorder", i);
              auto BCtarget = verts.template pack<3>("BCtarget", i);
              auto BCbasis = verts.template pack<3, 3>("BCbasis", i);
              auto x = verts.pack<3>("x", i);
              auto v = verts.pack<3>("v", i);
              vtemp("BCorder", voffset + i) = verts("BCorder", i);
              vtemp.template tuple<3>("BCtarget", voffset + i) = BCtarget;
              vtemp.template tuple<9>("BCbasis", voffset + i) = BCbasis;
              vtemp("BCfixed", voffset + i) = verts("BCfixed", i);

              vtemp("ws", voffset + i) = zs::sqrt(verts("m", i));
              vtemp.tuple<3>("xtilde", voffset + i) = x + v * dt;
              vtemp.tuple<3>("lambda", voffset + i) = vec3::zeros();
              vtemp.tuple<3>("xn", voffset + i) = x;
              if (BCorder > 0) {
                // recover original BCtarget
                BCtarget = BCbasis * BCtarget;
                vtemp.tuple<3>("vn", voffset + i) = (BCtarget - x) / dt;
              } else {
                vtemp.tuple<3>("vn", voffset + i) = v;
              }
              vtemp.tuple<3>("xt", voffset + i) = x;
              vtemp.tuple<3>("x0", voffset + i) = verts.pack<3>("x0", i);
            });
        // record surface (tri) indices
        auto &tris = primHandle.getSurfTris();
        pol(Collapse(tris.size()),
            [stInds = proxy<space>({}, stInds), tris = proxy<space>({}, tris),
             voffset = primHandle.vOffset,
             sfoffset = primHandle.sfOffset] __device__(int i) mutable {
              stInds.template tuple<3>("inds", sfoffset + i) =
                  (tris.template pack<3>("inds", i)
                       .template reinterpret_bits<int>() +
                   (int)voffset)
                      .template reinterpret_bits<float>();
            });
        auto &edges = primHandle.getSurfEdges();
        pol(Collapse(edges.size()),
            [seInds = proxy<space>({}, seInds), edges = proxy<space>({}, edges),
             voffset = primHandle.vOffset,
             seoffset = primHandle.seOffset] __device__(int i) mutable {
              seInds.template tuple<2>("inds", seoffset + i) =
                  (edges.template pack<2>("inds", i)
                       .template reinterpret_bits<int>() +
                   (int)voffset)
                      .template reinterpret_bits<float>();
            });
        auto &points = primHandle.getSurfVerts();
        pol(Collapse(points.size()),
            [svInds = proxy<space>({}, svInds),
             points = proxy<space>({}, points), voffset = primHandle.vOffset,
             svoffset = primHandle.svOffset] __device__(int i) mutable {
              svInds("inds", svoffset + i) = reinterpret_bits<float>(
                  reinterpret_bits<int>(points("inds", i)) + (int)voffset);
            });
      }
      pol(Collapse(coVerts.size()),
          [vtemp = proxy<space>({}, vtemp), coverts = proxy<space>({}, coVerts),
           coOffset = coOffset, dt = dt,
           augLagCoeff = augLagCoeff] __device__(int i) mutable {
            auto x = coverts.pack<3>("x", i);
            vec3 newX{};
            if (coverts.hasProperty("BCtarget"))
              newX = coverts.pack<3>("BCtarget", i);
            else {
              auto v = coverts.pack<3>("v", i);
              newX = x + v * dt;
            }
            vtemp("BCorder", coOffset + i) = 3;
            vtemp.template tuple<9>("BCbasis", coOffset + i) = mat3::identity();
            vtemp.template tuple<3>("BCtarget", coOffset + i) = newX;
            vtemp("BCfixed", coOffset + i) =
                (newX - x).l2NormSqr() == 0 ? 1 : 0;

            vtemp("ws", coOffset + i) = zs::sqrt(coverts("m", i) * augLagCoeff);
            vtemp.tuple<3>("xtilde", coOffset + i) = newX;
            vtemp.tuple<3>("lambda", coOffset + i) = vec3::zeros();
            vtemp.tuple<3>("xn", coOffset + i) = x;
            vtemp.tuple<3>("vn", coOffset + i) = (newX - x) / dt;
            vtemp.tuple<3>("xt", coOffset + i) = x;
            vtemp.tuple<3>("x0", coOffset + i) = coverts.pack<3>("x0", i);
          });
    }
    void advanceSubstep(zs::CudaExecutionPolicy &pol, T ratio) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      // setup substep dt
      dt = framedt * ratio;
      curRatio += ratio;
      pol(Collapse(coOffset),
          [vtemp = proxy<space>({}, vtemp), coOffset = coOffset, dt = dt, ratio,
           localRatio =
               ratio / (1 - curRatio + ratio)] __device__(int vi) mutable {
            int BCorder = vtemp("BCorder", vi);
            auto BCbasis = vtemp.pack<3, 3>("BCbasis", vi);
            auto projVec = [&BCbasis, BCorder](auto &dx) {
              dx = BCbasis.transpose() * dx;
              for (int d = 0; d != BCorder; ++d)
                dx[d] = 0;
              dx = BCbasis * dx;
            };
            auto xn = vtemp.template pack<3>("xn", vi);
            auto deltaX = vtemp.template pack<3>("vn", vi) * dt;
            if (BCorder > 0)
              projVec(deltaX);
            auto newX = xn + deltaX;
            vtemp.template tuple<3>("xtilde", vi) = newX;

            // update "BCfixed", "BCtarget" for dofs under boundary influence
            if (BCorder > 0) {
              vtemp.template tuple<3>("BCtarget", vi) =
                  BCbasis.transpose() * newX;
              vtemp("BCfixed", vi) = deltaX.l2NormSqr() == 0 ? 1 : 0;
            }
          });
      pol(Collapse(coVerts.size()),
          [vtemp = proxy<space>({}, vtemp), coverts = proxy<space>({}, coVerts),
           coOffset = coOffset, framedt = framedt,
           curRatio = curRatio] __device__(int i) mutable {
            auto xt = vtemp.template pack<3>("xt", coOffset + i);
            auto xn = vtemp.template pack<3>("xn", coOffset + i);
            vec3 newX{};
            if (coverts.hasProperty("BCtarget"))
              newX = coverts.pack<3>("BCtarget", i);
            else {
              auto v = coverts.pack<3>("v", i);
              newX = xt + v * framedt;
            }
            // auto xk = xt + (newX - xt) * curRatio;
            auto xk = newX * curRatio + (1 - curRatio) * xt;
            vtemp.template tuple<3>("BCtarget", coOffset + i) = xk;
            vtemp("BCfixed", coOffset + i) = (xk - xn).l2NormSqr() == 0 ? 1 : 0;
            vtemp.template tuple<3>("xtilde", coOffset + i) = xk;
          });
    }
    void updateVelocities(zs::CudaExecutionPolicy &pol) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      pol(zs::range(coOffset), [vtemp = proxy<space>({}, vtemp),
                                dt = dt] __device__(int vi) mutable {
        auto newX = vtemp.pack<3>("xn", vi);
        auto dv = (newX - vtemp.pack<3>("xtilde", vi)) / dt;
        auto vn = vtemp.pack<3>("vn", vi);
        vn += dv;
        int BCorder = vtemp("BCorder", vi);
        auto BCbasis = vtemp.pack<3, 3>("BCbasis", vi);
        auto projVec = [&BCbasis, BCorder](auto &dx) {
          dx = BCbasis.transpose() * dx;
          for (int d = 0; d != BCorder; ++d)
            dx[d] = 0;
          dx = BCbasis * dx;
        };
        if (BCorder > 0)
          projVec(vn);
        vtemp.tuple<3>("vn", vi) = vn;
      });
    }
    void updatePositionsAndVelocities(zs::CudaExecutionPolicy &pol) {
      using namespace zs;
      constexpr auto space = execspace_e::cuda;
      for (auto &primHandle : prims) {
        auto &verts = primHandle.getVerts();
        // update velocity and positions
        pol(zs::range(verts.size()),
            [vtemp = proxy<space>({}, vtemp), verts = proxy<space>({}, verts),
             dt = dt, vOffset = primHandle.vOffset] __device__(int vi) mutable {
              verts.tuple<3>("x", vi) = vtemp.pack<3>("xn", vOffset + vi);
              verts.tuple<3>("v", vi) = vtemp.pack<3>("vn", vOffset + vi);
            });
      }
    }

    IPCSystem(std::vector<ZenoParticles *> zsprims, const dtiles_t &coVerts,
              const tiles_t &coEdges, const tiles_t &coEles, T dt,
              const ZenoConstitutiveModel &models)
        : coVerts{coVerts}, coEdges{coEdges}, coEles{coEles},
          PP{estNumCps, zs::memsrc_e::um, 0},
          nPP{zsprims[0]->getParticles<true>().get_allocator(), 1},
          tempPP{{{"H", 36}, {"inds_pre", 2}, {"dist2_pre", 1}},
                 estNumCps,
                 zs::memsrc_e::um,
                 0},
          PE{estNumCps, zs::memsrc_e::um, 0},
          nPE{zsprims[0]->getParticles<true>().get_allocator(), 1},
          tempPE{{{"H", 81}, {"inds_pre", 3}, {"dist2_pre", 1}},
                 estNumCps,
                 zs::memsrc_e::um,
                 0},
          PT{estNumCps, zs::memsrc_e::um, 0},
          nPT{zsprims[0]->getParticles<true>().get_allocator(), 1},
          tempPT{{{"H", 144}, {"inds_pre", 4}, {"dist2_pre", 1}},
                 estNumCps,
                 zs::memsrc_e::um,
                 0},
          EE{estNumCps, zs::memsrc_e::um, 0},
          nEE{zsprims[0]->getParticles<true>().get_allocator(), 1},
          tempEE{{{"H", 144}, {"inds_pre", 4}, {"dist2_pre", 1}},
                 estNumCps,
                 zs::memsrc_e::um,
                 0},
          temp{5000000, zs::memsrc_e::um,
               zsprims[0]->getParticles<true>().devid()},
          csPT{estNumCps, zs::memsrc_e::um, 0}, csEE{estNumCps,
                                                     zs::memsrc_e::um, 0},
          ncsPT{zsprims[0]->getParticles<true>().get_allocator(), 1},
          ncsEE{zsprims[0]->getParticles<true>().get_allocator(), 1}, dt{dt},
          framedt{dt}, curRatio{0}, models{models} {
      coOffset = sfOffset = seOffset = svOffset = 0;
      prevNumPP = prevNumPE = prevNumPT = prevNumEE = 0;
      for (auto primPtr : zsprims) {
        if (primPtr->category == ZenoParticles::category_e::surface)
          prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset,
                             zs::wrapv<3>{});
        else if (primPtr->category == ZenoParticles::category_e::tet)
          prims.emplace_back(*primPtr, coOffset, sfOffset, seOffset, svOffset,
                             zs::wrapv<4>{});
      }
      numDofs = coOffset + coVerts.size();
      vtemp = dtiles_t{
          zsprims[0]->getParticles<true>().get_allocator(),
          {{"grad", 3},
           {"P", 9},
           // dirichlet boundary condition type; 0: NOT, 1: ZERO, 2: NONZERO
           {"BCorder", 1},
           {"BCbasis", 9},
           {"BCtarget", 3},
           {"BCfixed", 1},
           {"ws", 1}, // also as constraint jacobian
           {"cons", 3},
           {"lambda", 3},

           {"dir", 3},
           {"xn", 3},
           {"vn", 3},
           {"x0", 3}, // initial positions
           {"xt", 3}, // initial positions at the current timestep
           {"xn0", 3},
           {"xtilde", 3},
           {"temp", 3},
           {"r", 3},
           {"p", 3},
           {"q", 3}},
          numDofs};
      // ground + inertial hessian
      tempPB = dtiles_t{vtemp.get_allocator(), {{"H", 9}, {"Hi", 9}}, coOffset};
      nPP.setVal(0);
      nPE.setVal(0);
      nPT.setVal(0);
      nEE.setVal(0);

      ncsPT.setVal(0);
      ncsEE.setVal(0);

      auto cudaPol = zs::cuda_exec();
      // average edge length (for CCD filtering)
      meanEdgeLength = averageSurfEdgeLength(cudaPol);
      initialize(cudaPol);
      fmt::print("num total obj <verts, surfV, surfE, surfT>: {}, {}, {}, {}\n",
                 coOffset, svOffset, seOffset, sfOffset);
      {
        {
          auto triBvs = retrieve_bounding_volumes(cudaPol, vtemp, "xn", stInds,
                                                  zs::wrapv<3>{}, 0);
          stBvh.build(cudaPol, triBvs);
          auto edgeBvs = retrieve_bounding_volumes(cudaPol, vtemp, "xn", seInds,
                                                   zs::wrapv<2>{}, 0);
          seBvh.build(cudaPol, edgeBvs);
        }
        {
          auto triBvs = retrieve_bounding_volumes(cudaPol, vtemp, "xn", coEles,
                                                  zs::wrapv<3>{}, coOffset);
          bouStBvh.build(cudaPol, triBvs);
          auto edgeBvs = retrieve_bounding_volumes(
              cudaPol, vtemp, "xn", coEdges, zs::wrapv<2>{}, coOffset);
          bouSeBvh.build(cudaPol, edgeBvs);
        }
      }
    }

    std::vector<PrimitiveHandle> prims;

    // (scripted) collision objects
    const dtiles_t &coVerts;
    const tiles_t &coEdges, &coEles;
    dtiles_t vtemp;
    // self contacts
    using pair_t = zs::vec<int, 2>;
    using pair3_t = zs::vec<int, 3>;
    using pair4_t = zs::vec<int, 4>;
    using dpair_t = zs::vec<Ti, 2>;
    using dpair3_t = zs::vec<Ti, 3>;
    using dpair4_t = zs::vec<Ti, 4>;
    zs::Vector<pair_t> PP;
    zs::Vector<int> nPP;
    dtiles_t tempPP;
    zs::Vector<pair3_t> PE;
    zs::Vector<int> nPE;
    dtiles_t tempPE;
    zs::Vector<pair4_t> PT;
    zs::Vector<int> nPT;
    dtiles_t tempPT;
    zs::Vector<pair4_t> EE;
    zs::Vector<int> nEE;
    dtiles_t tempEE;

    zs::Vector<T> temp;

    int prevNumPP, prevNumPE, prevNumPT, prevNumEE;

    zs::Vector<pair4_t> csPT, csEE;
    zs::Vector<int> ncsPT, ncsEE;

    // boundary contacts
    dtiles_t tempPB;
    // end contacts
    const ZenoConstitutiveModel &models;
    // auxiliary data (spatial acceleration)
    bvh_t stBvh, seBvh; // for simulated objects
    tiles_t stInds, seInds, svInds;
    std::size_t coOffset, numDofs;
    std::size_t sfOffset, seOffset, svOffset;
    bvh_t bouStBvh, bouSeBvh; // for collision objects
    T meanEdgeLength, dt, framedt, curRatio;
  };

  void apply() override {
    using namespace zs;
    constexpr auto space = execspace_e::cuda;
    auto cudaPol = cuda_exec().sync(true);

    auto zstets = RETRIEVE_OBJECT_PTRS(ZenoParticles, "ZSParticles");
    // auto zstets = get_input<ZenoParticles>("ZSParticles");
    std::shared_ptr<ZenoParticles> zsboundary;
    if (has_input<ZenoParticles>("ZSBoundaryPrimitives"))
      zsboundary = get_input<ZenoParticles>("ZSBoundaryPrimitives");
    auto models = zstets[0]->getModel();
    auto dt = get_input2<float>("dt");

    /// solver parameters
    auto input_est_num_cps = get_input2<int>("est_num_cps");
    auto input_dHat = get_input2<float>("dHat");
    auto input_kappa0 = get_input2<float>("kappa0");
    auto input_aug_coeff = get_input2<float>("aug_coeff");
    auto input_pn_rel = get_input2<float>("pn_rel");
    auto input_cg_rel = get_input2<float>("cg_rel");
    auto input_gravity = get_input2<float>("gravity");
    auto input_pn_cap = get_input2<int>("pn_iter_cap");
    auto input_cg_cap = get_input2<int>("cg_iter_cap");
    auto input_ccd_cap = get_input2<int>("ccd_iter_cap");

    int nSubsteps = get_input2<int>("num_substeps");

    kappa0 = input_kappa0;
    augLagCoeff = input_aug_coeff;
    pnRel = input_pn_rel;
    cgRel = input_cg_rel;
    PNCap = input_pn_cap;
    CGCap = input_cg_cap;
    CCDCap = input_ccd_cap;

    /// if there are no high precision verts, init from the low precision one
    for (auto zstet : zstets) {
      if (!zstet->hasImage(ZenoParticles::s_particleTag)) {
        auto &loVerts = zstet->getParticles();
        auto &verts = zstet->images[ZenoParticles::s_particleTag];
        verts = typename ZenoParticles::dtiles_t{
            loVerts.get_allocator(), loVerts.getPropertyTags(), loVerts.size()};
        cudaPol(range(verts.size()),
                [loVerts = proxy<space>({}, loVerts),
                 verts = proxy<space>({}, verts)] __device__(int vi) mutable {
                  // make sure there are no "inds"-like properties in verts!
                  for (int propid = 0; propid != verts._N; ++propid) {
                    auto propOffset = verts._tagOffsets[propid];
                    for (int chn = 0; chn != verts._tagSizes[propid]; ++chn)
                      verts(propOffset + chn, vi) =
                          loVerts(propOffset + chn, vi);
                  }
                });
      }
    }
    if (!zsboundary->hasImage(ZenoParticles::s_particleTag)) {
      auto &loVerts = zsboundary->getParticles();
      auto &verts = zsboundary->images[ZenoParticles::s_particleTag];
      verts = typename ZenoParticles::dtiles_t{
          loVerts.get_allocator(), loVerts.getPropertyTags(), loVerts.size()};
      cudaPol(range(verts.size()),
              [loVerts = proxy<space>({}, loVerts),
               verts = proxy<space>({}, verts)] __device__(int vi) mutable {
                // make sure there are no "inds"-like properties in verts!
                for (int propid = 0; propid != verts._N; ++propid) {
                  auto propOffset = verts._tagOffsets[propid];
                  for (int chn = 0; chn != verts._tagSizes[propid]; ++chn)
                    verts(propOffset + chn, vi) = loVerts(propOffset + chn, vi);
                }
              });
    }
    const dtiles_t &coVerts =
        zsboundary ? zsboundary->getParticles<true>() : dtiles_t{};
    const tiles_t &coEdges =
        zsboundary ? (*zsboundary)[ZenoParticles::s_surfEdgeTag] : tiles_t{};
    const tiles_t &coEles =
        zsboundary ? zsboundary->getQuadraturePoints() : tiles_t{};

    IPCSystem A{zstets, coVerts, coEdges, coEles, dt, models};

    auto coOffset = A.coOffset;
    auto numDofs = A.numDofs;

    estNumCps = input_est_num_cps > 0
                    ? input_est_num_cps // if specified, overwrite
                    : std::max(numDofs * 4, estNumCps);

    dtiles_t &vtemp = A.vtemp;

    /// time integrator
    dHat = input_dHat;
    extForce = vec3{0, input_gravity, 0};
    kappa = kappa0;
    targetGRes = pnRel;

#if s_enableAdaptiveSetting
    {
      A.updateWholeBoundingBoxSize(cudaPol);
      fmt::print("box diag size: {}\n", std::sqrt(boxDiagSize2));
      /// dHat
      dHat = input_dHat * std::sqrt(boxDiagSize2);
      /// grad pn residual tolerance
      targetGRes = pnRel * std::sqrt(boxDiagSize2);
      /// mean mass
      avgNodeMass = 0;
      T sumNodeMass = 0;
      int sumNodes = 0;
      Vector<T> masses{vtemp.get_allocator(), coOffset};
      cudaPol(Collapse{coOffset},
              [masses = proxy<space>(masses),
               vtemp = proxy<space>({}, vtemp)] __device__(int vi) mutable {
                masses[vi] = zs::sqr(vtemp("ws", vi));
              });
      auto tmp = A.reduce(cudaPol, masses);
      sumNodeMass += tmp;
      sumNodes = coOffset;
      avgNodeMass = sumNodeMass / sumNodes;
      /// kappaMin
      A.initKappa(cudaPol);
      /// adaptive kappa
      {
        T H_b = computeHb((T)1e-16 * boxDiagSize2, dHat * dHat);
        kappa = 1e11 * avgNodeMass / (4e-16 * boxDiagSize2 * H_b);
        kappaMax = 100 * kappa;
        if (kappa < kappaMin)
          kappa = kappaMin;
        if (kappa > kappaMax)
          kappa = kappaMax;
      }
      fmt::print("auto dHat: {}, targetGRes: {}\n", dHat, targetGRes);
      fmt::print("average node mass: {}, kappa: {} ({} - {})\n", avgNodeMass,
                 kappa, kappaMin, kappaMax);
      // getchar();
    }
#endif

    // nSubsteps = 1;
    for (int subi = 0; subi != nSubsteps; ++subi) {
      fmt::print("processing substep {}\n", subi);

      projectDBC = false;
      BCsatisfied = false;
      useGD = false;
      A.advanceSubstep(cudaPol, (T)1 / nSubsteps);

      /// optimizer
      for (int newtonIter = 0; newtonIter != PNCap; ++newtonIter) {
        // check constraints
        if (!BCsatisfied) {
          A.computeConstraints(cudaPol, "xn");
          auto cr = A.constraintResidual(cudaPol, true);
          if (A.areConstraintsSatisfied(cudaPol)) {
            fmt::print("satisfied cons res [{}] at newton iter [{}]\n", cr,
                       newtonIter);
            // A.checkDBCStatus(cudaPol);
            // getchar();
            projectDBC = true;
            BCsatisfied = true;
          }
          fmt::print(fg(fmt::color::alice_blue),
                     "substep {} newton iter {} cons residual: {}\n", subi,
                     newtonIter, cr);
        }

        if constexpr (s_enableContact)
          A.findCollisionConstraints(cudaPol, dHat, xi,
                                     s_enableAdaptiveSetting);
        // construct gradient, prepare hessian, prepare preconditioner
        cudaPol(zs::range(numDofs),
                [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
                  vtemp.tuple<9>("P", i) = mat3::zeros();
                  vtemp.tuple<3>("grad", i) = vec3::zeros();
                });
        A.computeInertialAndGravityPotentialGradient(cudaPol, "grad");
        match([&](auto &elasticModel) {
          A.computeElasticGradientAndHessian(cudaPol, elasticModel);
        })(models.getElasticModel());
        A.computeBoundaryBarrierGradientAndHessian(cudaPol);
        if constexpr (s_enableContact)
          A.computeBarrierGradientAndHessian(cudaPol);

        // rotate gradient and project
        A.project(cudaPol, "grad");
        // apply constraints (augmented lagrangians) after rotation!
        if (!BCsatisfied) {
          // grad
          cudaPol(zs::range(numDofs),
                  [vtemp = proxy<space>({}, vtemp),
                   kappa = kappa] __device__(int i) mutable {
                    // computed during the previous constraint residual check
                    auto cons = vtemp.pack<3>("cons", i);
                    auto w = vtemp("ws", i);
                    vtemp.tuple<3>("grad", i) = vtemp.pack<3>("grad", i) +
                                                w * vtemp.pack<3>("lambda", i) -
                                                kappa * w * cons;
                    for (int d = 0; d != 3; ++d)
                      if (cons[d] != 0) {
                        vtemp("P", 4 * d, i) += kappa * w;
                      }
                  });
          // hess (embedded in multiply)
        }

        // prepare preconditioner
        cudaPol(zs::range(coVerts.size()),
                [vtemp = proxy<space>({}, vtemp), coOffset,
                 kappa = kappa] __device__(int i) mutable {
                  auto cons = vtemp.pack<3>("cons", i);
                  auto w = vtemp("ws", coOffset + i);
                  if (cons.l2NormSqr() != 0)
                    vtemp.tuple<9>("P", coOffset + i) =
                        mat3::identity() * kappa * w;
#if 0
                int d = 0;
                for (; d != 3 && cons[d] != 0; ++d)
                  ;
                for (; d != 3; ++d)
                  vtemp("P", 4 * d, coOffset + i) = kappa * w;
#endif
                });
        cudaPol(zs::range(numDofs),
                [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
                  auto mat = vtemp.pack<3, 3>("P", i);
                  if (zs::abs(zs::determinant(mat)) > limits<T>::epsilon() * 10)
                    vtemp.tuple<9>("P", i) = inverse(mat);
                  else
                    vtemp.tuple<9>("P", i) = mat3::identity();
                });

        // modify initial x so that it satisfied the constraint.

        // A dir = grad
        A.cgsolve(cudaPol, useGD);

        // recover rotated solution
        cudaPol(Collapse{vtemp.size()},
                [vtemp = proxy<space>({}, vtemp)] __device__(int vi) mutable {
                  vtemp.tuple<3>("dir", vi) = vtemp.pack<3, 3>("BCbasis", vi) *
                                              vtemp.pack<3>("dir", vi);
                });
        // check "dir" inf norm
        T res = A.infNorm(cudaPol, vtemp, "dir") / dt;
        T cons_res = A.constraintResidual(cudaPol);
        if (!useGD && res < targetGRes && cons_res == 0) {
          fmt::print("\t# newton optimizer ends in {} iters with residual {}\n",
                     newtonIter, res);
          break;
        }

        fmt::print(fg(fmt::color::aquamarine),
                   "substep {} newton iter {}: direction residual(/dt) {}, "
                   "grad residual {}\n",
                   subi, newtonIter, res, A.infNorm(cudaPol, vtemp, "grad"));

        // xn0 <- xn for line search
        cudaPol(zs::range(vtemp.size()),
                [vtemp = proxy<space>({}, vtemp)] __device__(int i) mutable {
                  vtemp.tuple<3>("xn0", i) = vtemp.pack<3>("xn", i);
                });

        // line search
        bool CCDfiltered = false;
        T alpha = 1.;
        T prevAlpha{limits<T>::infinity()};
        { //
#if 1
          // average length
          zs::Vector<T> lens{vtemp.get_allocator(), coOffset};
          cudaPol(Collapse{coOffset},
                  [lens = proxy<space>(lens),
                   vtemp = proxy<space>({}, vtemp)] __device__(int ei) mutable {
                    // lens[ei] = vtemp.template pack<3>("dir", ei).norm();
                    lens[ei] = vtemp.template pack<3>("dir", ei).abs().sum();
                  });
          auto meanDirSize = (A.reduce(cudaPol, lens) / dt) / coOffset;
          auto spanSize =
              meanDirSize * alpha / (A.meanEdgeLength * refStepsizeCoeff);
#else
          // infNorm
          auto spanSize = res * alpha / (A.meanEdgeLength * 1);
#endif
          if (spanSize > 1) { // mainly for reducing ccd pairs
            alpha /= spanSize;
            CCDfiltered = true;
            fmt::print("\tstepsize after dir magnitude pre-filtering: {} "
                       "(spansize: {})\n",
                       alpha, spanSize);
            prevAlpha = alpha;
          }
        }
        A.groundIntersectionFreeStepsize(cudaPol, alpha);
        fmt::print("\tstepsize after ground: {}\n", alpha);
#if s_enableContact
        {
          // A.intersectionFreeStepsize(cudaPol, xi, alpha);
          // fmt::print("\tstepsize after intersection-free: {}\n", alpha);
          A.findCCDConstraints(cudaPol, alpha, xi);
          auto [npp, npe, npt, nee, ncspt, ncsee] = A.getCnts();
          A.intersectionFreeStepsize(cudaPol, xi, alpha);
          fmt::print("\tstepsize after ccd: {}. (ncspt: {}, ncsee: {})\n",
                     alpha, ncspt, ncsee);
/// check discrete collision
#if s_enableDCDCheck
          while (A.checkSelfIntersection(cudaPol)) {
            alpha /= 2;
            cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp),
                                              alpha] __device__(int i) mutable {
              vtemp.tuple<3>("xn", i) =
                  vtemp.pack<3>("xn0", i) + alpha * vtemp.pack<3>("dir", i);
            });
          }
#endif
        }
#endif

        A.lineSearch(cudaPol, alpha, CCDfiltered);

        if (CCDfiltered && prevAlpha == alpha) {
          if (++numContinuousCap < 8)
            refStepsizeCoeff++;
        } else {
          refStepsizeCoeff = 1;
          numContinuousCap = 0;
        }

        cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp),
                                          alpha] __device__(int i) mutable {
          vtemp.tuple<3>("xn", i) =
              vtemp.pack<3>("xn0", i) + alpha * vtemp.pack<3>("dir", i);
        });

/// check discrete collision
#if s_enableDCDCheck
        while (A.checkSelfIntersection(cudaPol)) {
          alpha /= 2;
          cudaPol(zs::range(vtemp.size()), [vtemp = proxy<space>({}, vtemp),
                                            alpha] __device__(int i) mutable {
            vtemp.tuple<3>("xn", i) =
                vtemp.pack<3>("xn0", i) + alpha * vtemp.pack<3>("dir", i);
          });
        }
#endif

        if (alpha < 1e-8) {
          useGD = true;
        } else {
          useGD = false;
        }

#if s_enableAdaptiveSetting
        if (A.updateKappaRequired(cudaPol))
          if (kappa < kappaMax) {
            kappa *= 2;
            fmt::print(fg(fmt::color::alice_blue),
                       "increasing kappa to {} (max: {})\n", kappa, kappaMax);
          }
#endif

        // update rule
        cons_res = A.constraintResidual(cudaPol);
        if (res < updateZoneTol && cons_res > consTol) {
          if (kappa < kappaMax)
            kappa *= 2;
          else {
            cudaPol(Collapse{numDofs},
                    [vtemp = proxy<space>({}, vtemp),
                     kappa = kappa] __device__(int vi) mutable {
                      if (int BCorder = vtemp("BCorder", vi); BCorder > 0) {
                        vtemp.tuple<3>("lambda", vi) =
                            vtemp.pack<3>("lambda", vi) -
                            kappa * vtemp("ws", vi) * vtemp.pack<3>("cons", vi);
                      }
                    });
          }
        }
      } // end newton step

      A.updateVelocities(cudaPol);
    }

    // update velocity and positions
    A.updatePositionsAndVelocities(cudaPol);
    // not sure if this is necessary for numerical reasons
    if (coVerts.size())
      cudaPol(zs::range(coVerts.size()),
              [vtemp = proxy<space>({}, vtemp),
               verts = proxy<space>({}, zsboundary->getParticles<true>()),
               loVerts = proxy<space>({}, zsboundary->getParticles()),
               coOffset] __device__(int vi) mutable {
                auto newX = vtemp.pack<3>("xn", coOffset + vi);
                verts.tuple<3>("x", vi) = newX;
                loVerts.tuple<3>("x", vi) = newX;
                // no need to update v here. positions are moved accordingly
                // also, boundary velocies are set elsewhere
              });

    set_output("ZSParticles", get_input("ZSParticles"));
  }
};

ZENDEFNODE(CodimStepping, {{
                               "ZSParticles",
                               "ZSBoundaryPrimitives",
                               {"int", "est_num_cps", "0"},
                               {"float", "dt", "0.01"},
                               {"float", "dHat", "0.001"},
                               {"float", "kappa0", "1e3"},
                               {"float", "aug_coeff", "1e3"},
                               {"float", "pn_rel", "0.01"},
                               {"float", "cg_rel", "0.0001"},
                               {"int", "pn_iter_cap", "1000"},
                               {"int", "cg_iter_cap", "500"},
                               {"int", "ccd_iter_cap", "20000"},
                               {"float", "gravity", "-9.0"},
                               {"int", "num_substeps", "1"},
                           },
                           {"ZSParticles"},
                           {},
                           {"FEM"}});

} // namespace zeno

#include "Ipc.inl"